#include "hip/hip_runtime.h"
#include "cuda/constant_mem.h"
#include "cuda/constant_mem_func.h"
#include "cuda/cuda_utility.h"
#include "field_solver.h"
#include "interpolation.h"
#include "utils/timer.h"
#include "utils/nvproftool.h"

#define BLOCK_SIZE_X 32
#define BLOCK_SIZE_Y 4
#define BLOCK_SIZE_Z 4

#define SHIFT_GHOST 2

#define TINY 1e-7

namespace Coffee {

static dim3 blockSize(BLOCK_SIZE_X, BLOCK_SIZE_Y, BLOCK_SIZE_Z);

static dim3 blockGroupSize;

__global__ void
kernel_compute_rho_thread(const Scalar *ex, const Scalar *ey, const Scalar *ez,
                          Scalar *rho, int shift) {
  size_t ijk;
  int i = threadIdx.x + blockIdx.x * blockDim.x + dev_grid.guard[0] - shift;
  int j = threadIdx.y + blockIdx.y * blockDim.y + dev_grid.guard[1] - shift;
  int k = threadIdx.z + blockIdx.z * blockDim.z + dev_grid.guard[2] - shift;
  if (i < dev_grid.dims[0] - dev_grid.guard[0] + shift &&
      j < dev_grid.dims[1] - dev_grid.guard[1] + shift &&
      k < dev_grid.dims[2] - dev_grid.guard[2] + shift) {
    ijk = i + j * dev_grid.dims[0] +
          k * dev_grid.dims[0] * dev_grid.dims[1];
    rho[ijk] = dev_grid.inv_delta[0] * (ex[ijk] - ex[ijk - 1]) +
               dev_grid.inv_delta[1] * (ey[ijk] - ey[ijk - dev_grid.dims[0]]) +
               dev_grid.inv_delta[2] * (ez[ijk] - ez[ijk - dev_grid.dims[0] * dev_grid.dims[1]]);
  }
}

__global__ void
kernel_rk_push_thread(const Scalar *ex, const Scalar *ey, const Scalar *ez,
                      const Scalar *bx, const Scalar *by, const Scalar *bz,
                      const Scalar *bx0, const Scalar *by0,
                      const Scalar *bz0, Scalar *dex, Scalar *dey,
                      Scalar *dez, Scalar *dbx, Scalar *dby, Scalar *dbz,
                      Scalar *rho, int shift) {
  Scalar CCx = dev_params.dt * dev_grid.inv_delta[0];
  Scalar CCy = dev_params.dt * dev_grid.inv_delta[1];
  Scalar CCz = dev_params.dt * dev_grid.inv_delta[2];
  Scalar intex, intey, intez, intbx, intby, intbz, intrho;
  Scalar jx, jy, jz;
  size_t ijk, iP1jk, iM1jk, ijP1k, ijM1k, ijkP1, ijkM1;

  int i = threadIdx.x + blockIdx.x * blockDim.x + dev_grid.guard[0] - shift;
  int j = threadIdx.y + blockIdx.y * blockDim.y + dev_grid.guard[1] - shift;
  int k = threadIdx.z + blockIdx.z * blockDim.z + dev_grid.guard[2] - shift;
  if (i < dev_grid.dims[0] - dev_grid.guard[0] + shift &&
      j < dev_grid.dims[1] - dev_grid.guard[1] + shift &&
      k < dev_grid.dims[2] - dev_grid.guard[2] + shift) {
    ijk = i + j * dev_grid.dims[0] +
          k * dev_grid.dims[0] * dev_grid.dims[1];
    iP1jk = ijk + 1;
    iM1jk = ijk - 1;
    ijP1k = ijk + dev_grid.dims[0];
    ijM1k = ijk - dev_grid.dims[0];
    ijkP1 = ijk + dev_grid.dims[0] * dev_grid.dims[1];
    ijkM1 = ijk - dev_grid.dims[0] * dev_grid.dims[1];
    // push B-field
    dbx[ijk] = CCx * (ey[ijkP1] - ey[ijk] - ez[ijP1k] + ez[ijk]);
    dby[ijk] = CCy * (ez[iP1jk] - ez[ijk] - ex[ijkP1] + ex[ijk]);
    dbz[ijk] = CCz * (ex[ijP1k] - ex[ijk] - ey[iP1jk] + ey[ijk]);
    // push E-field
    dex[ijk] = CCx * ((by[ijkM1] - by[ijk] - bz[ijM1k] + bz[ijk]) -
                      (by0[ijkM1] - bz0[ijk] - bz0[ijM1k] + bz0[ijk]));
    dey[ijk] = CCy * ((bz[iM1jk] - bz[ijk] - bx[ijkM1] + bx[ijk]) -
                      (bz0[iM1jk] - bz0[ijk] - bx0[ijkM1] + bx0[ijk]));
    dez[ijk] = CCz * ((bx[ijM1k] - bx[ijk] - by[iM1jk] + by[ijk]) -
                      (bx0[ijM1k] - bx0[ijk] - by0[iM1jk] + by0[ijk]));

    // computing currents
    //   `j_x`:
    intrho = interpolate(rho, ijk, Stagger(0b111), Stagger(0b110),
                         dev_grid.dims[0], dev_grid.dims[1]);
    intex = interpolate(ex, ijk, Stagger(0b110), Stagger(0b110),
                       dev_grid.dims[0], dev_grid.dims[1]);
    intey = interpolate(ey, ijk, Stagger(0b101), Stagger(0b110),
                       dev_grid.dims[0], dev_grid.dims[1]);
    intez = interpolate(ez, ijk, Stagger(0b011), Stagger(0b110),
                       dev_grid.dims[0], dev_grid.dims[1]);
    intbx = interpolate(bx, ijk, Stagger(0b001), Stagger(0b110),
                       dev_grid.dims[0], dev_grid.dims[1]);
    intby = interpolate(by, ijk, Stagger(0b010), Stagger(0b110),
                       dev_grid.dims[0], dev_grid.dims[1]);
    intbz = interpolate(bz, ijk, Stagger(0b100), Stagger(0b110),
                       dev_grid.dims[0], dev_grid.dims[1]);
    jx = dev_params.dt * intrho * (intey * intbz - intby * intez) /
         (intbx * intbx + intby * intby + intbz * intbz + TINY);
    //   `j_y`:
    intrho = interpolate(rho, ijk, Stagger(0b111), Stagger(0b101),
                         dev_grid.dims[0], dev_grid.dims[1]);
    intex = interpolate(ex, ijk, Stagger(0b110), Stagger(0b101),
                       dev_grid.dims[0], dev_grid.dims[1]);
    intey = interpolate(ey, ijk, Stagger(0b101), Stagger(0b101),
                       dev_grid.dims[0], dev_grid.dims[1]);
    intez = interpolate(ez, ijk, Stagger(0b011), Stagger(0b101),
                       dev_grid.dims[0], dev_grid.dims[1]);
    intbx = interpolate(bx, ijk, Stagger(0b001), Stagger(0b101),
                       dev_grid.dims[0], dev_grid.dims[1]);
    intby = interpolate(by, ijk, Stagger(0b010), Stagger(0b101),
                       dev_grid.dims[0], dev_grid.dims[1]);
    intbz = interpolate(bz, ijk, Stagger(0b100), Stagger(0b101),
                       dev_grid.dims[0], dev_grid.dims[1]);
    jy = dev_params.dt * intrho * (intez * intbx - intex * intbz) /
         (intbx * intbx + intby * intby + intbz * intbz + TINY);
    //   `j_z`:
    intrho = interpolate(rho, ijk, Stagger(0b111), Stagger(0b011),
                         dev_grid.dims[0], dev_grid.dims[1]);
    intex = interpolate(ex, ijk, Stagger(0b110), Stagger(0b011),
                       dev_grid.dims[0], dev_grid.dims[1]);
    intey = interpolate(ey, ijk, Stagger(0b101), Stagger(0b011),
                       dev_grid.dims[0], dev_grid.dims[1]);
    intez = interpolate(ez, ijk, Stagger(0b011), Stagger(0b011),
                       dev_grid.dims[0], dev_grid.dims[1]);
    intbx = interpolate(bx, ijk, Stagger(0b001), Stagger(0b011),
                       dev_grid.dims[0], dev_grid.dims[1]);
    intby = interpolate(by, ijk, Stagger(0b010), Stagger(0b011),
                       dev_grid.dims[0], dev_grid.dims[1]);
    intbz = interpolate(bz, ijk, Stagger(0b100), Stagger(0b011),
                       dev_grid.dims[0], dev_grid.dims[1]);
    jz = dev_params.dt * intrho * (intex * intby - intbx * intey) /
         (intbx * intbx + intby * intby + intbz * intbz + TINY);

    dex[ijk] -= jx;
    dey[ijk] -= jy;
    dez[ijk] -= jz;
  }
}

__global__ void
kernel_rk_update_thread(Scalar *ex, Scalar *ey, Scalar *ez, Scalar *bx,
                        Scalar *by, Scalar *bz, const Scalar *enx,
                        const Scalar *eny, const Scalar *enz,
                        const Scalar *bnx, const Scalar *bny,
                        const Scalar *bnz, Scalar *dex, Scalar *dey,
                        Scalar *dez, const Scalar *dbx, const Scalar *dby,
                        const Scalar *dbz, Scalar rk_c1, Scalar rk_c2,
                        Scalar rk_c3, int shift) {
  size_t ijk;
  int i = threadIdx.x + blockIdx.x * blockDim.x + dev_grid.guard[0] - shift;
  int j = threadIdx.y + blockIdx.y * blockDim.y + dev_grid.guard[1] - shift;
  int k = threadIdx.z + blockIdx.z * blockDim.z + dev_grid.guard[2] - shift;
  if (i < dev_grid.dims[0] - dev_grid.guard[0] + shift &&
      j < dev_grid.dims[1] - dev_grid.guard[1] + shift &&
      k < dev_grid.dims[2] - dev_grid.guard[2] + shift) {
    ijk = i + j * dev_grid.dims[0] +
          k * dev_grid.dims[0] * dev_grid.dims[1];
    // update E-field
    ex[ijk] = rk_c1 * enx[ijk] + rk_c2 * ex[ijk] + rk_c3 * dex[ijk];
    ey[ijk] = rk_c1 * eny[ijk] + rk_c2 * ey[ijk] + rk_c3 * dey[ijk];
    ez[ijk] = rk_c1 * enz[ijk] + rk_c2 * ez[ijk] + rk_c3 * dez[ijk];
    dex[ijk] = ex[ijk];
    dey[ijk] = ey[ijk];
    dez[ijk] = ez[ijk];
    // update B-field
    bx[ijk] = rk_c1 * bnx[ijk] + rk_c2 * bx[ijk] + rk_c3 * dbx[ijk];
    by[ijk] = rk_c1 * bny[ijk] + rk_c2 * by[ijk] + rk_c3 * dby[ijk];
    bz[ijk] = rk_c1 * bnz[ijk] + rk_c2 * bz[ijk] + rk_c3 * dbz[ijk];
  }
}

__global__ void
kernel_clean_epar_thread(const Scalar *ex, const Scalar *ey, const Scalar *ez,
                         const Scalar *bx, const Scalar *by, const Scalar *bz,
                         Scalar *dex, Scalar *dey, Scalar *dez, int shift) {
  Scalar intex, intey, intez, intbx, intby, intbz;
  size_t ijk;
  int i = threadIdx.x + blockIdx.x * blockDim.x + dev_grid.guard[0] - shift;
  int j = threadIdx.y + blockIdx.y * blockDim.y + dev_grid.guard[1] - shift;
  int k = threadIdx.z + blockIdx.z * blockDim.z + dev_grid.guard[2] - shift;
  if (i < dev_grid.dims[0] - dev_grid.guard[0] + shift &&
      j < dev_grid.dims[1] - dev_grid.guard[1] + shift &&
      k < dev_grid.dims[2] - dev_grid.guard[2] + shift) {
    ijk = i + j * dev_grid.dims[0] +
          k * dev_grid.dims[0] * dev_grid.dims[1];
    // x:
    intex = interpolate(ex, ijk, Stagger(0b110), Stagger(0b110),
                        dev_grid.dims[0], dev_grid.dims[1]);
    intey = interpolate(ey, ijk, Stagger(0b101), Stagger(0b110),
                        dev_grid.dims[0], dev_grid.dims[1]);
    intez = interpolate(ez, ijk, Stagger(0b011), Stagger(0b110),
                        dev_grid.dims[0], dev_grid.dims[1]);
    intbx = interpolate(bx, ijk, Stagger(0b001), Stagger(0b110),
                        dev_grid.dims[0], dev_grid.dims[1]);
    intby = interpolate(by, ijk, Stagger(0b010), Stagger(0b110),
                        dev_grid.dims[0], dev_grid.dims[1]);
    intbz = interpolate(bz, ijk, Stagger(0b100), Stagger(0b110),
                        dev_grid.dims[0], dev_grid.dims[1]);
    dex[ijk] = ex[ijk] -
               (intex * intbx + intey * intby + intez * intbz) *
                   intbx /
                   (intbx * intbx + intby * intby + intbz * intbz + TINY);

    // y:
    intex = interpolate(ex, ijk, Stagger(0b110), Stagger(0b101),
                        dev_grid.dims[0], dev_grid.dims[1]);
    intey = interpolate(ey, ijk, Stagger(0b101), Stagger(0b101),
                        dev_grid.dims[0], dev_grid.dims[1]);
    intez = interpolate(ez, ijk, Stagger(0b011), Stagger(0b101),
                        dev_grid.dims[0], dev_grid.dims[1]);
    intbx = interpolate(bx, ijk, Stagger(0b001), Stagger(0b101),
                        dev_grid.dims[0], dev_grid.dims[1]);
    intby = interpolate(by, ijk, Stagger(0b010), Stagger(0b101),
                        dev_grid.dims[0], dev_grid.dims[1]);
    intbz = interpolate(bz, ijk, Stagger(0b100), Stagger(0b101),
                        dev_grid.dims[0], dev_grid.dims[1]);
    dey[ijk] = ey[ijk] -
               (intex * intbx + intey * intby + intez * intbz) *
                   intby /
                   (intbx * intbx + intby * intby + intbz * intbz + TINY);

    // z:
    intex = interpolate(ex, ijk, Stagger(0b110), Stagger(0b011),
                        dev_grid.dims[0], dev_grid.dims[1]);
    intey = interpolate(ey, ijk, Stagger(0b101), Stagger(0b011),
                        dev_grid.dims[0], dev_grid.dims[1]);
    intez = interpolate(ez, ijk, Stagger(0b011), Stagger(0b011),
                        dev_grid.dims[0], dev_grid.dims[1]);
    intbx = interpolate(bx, ijk, Stagger(0b001), Stagger(0b011),
                        dev_grid.dims[0], dev_grid.dims[1]);
    intby = interpolate(by, ijk, Stagger(0b010), Stagger(0b011),
                        dev_grid.dims[0], dev_grid.dims[1]);
    intbz = interpolate(bz, ijk, Stagger(0b100), Stagger(0b011),
                        dev_grid.dims[0], dev_grid.dims[1]);
    dez[ijk] = ez[ijk] -
               (intex * intbx + intey * intby + intez * intbz) *
                   intbz /
                   (intbx * intbx + intby * intby + intbz * intbz + TINY);
  }
}

__global__ void
kernel_check_eGTb_thread(const Scalar *dex, const Scalar *dey,
                         const Scalar *dez, Scalar *ex, Scalar *ey, Scalar *ez,
                         const Scalar *bx, const Scalar *by,
                         const Scalar *bz, int shift) {
  Scalar intex, intey, intez, intbx, intby, intbz, emag, bmag, temp;
  size_t ijk;
  int i = threadIdx.x + blockIdx.x * blockDim.x + dev_grid.guard[0] - shift;
  int j = threadIdx.y + blockIdx.y * blockDim.y + dev_grid.guard[1] - shift;
  int k = threadIdx.z + blockIdx.z * blockDim.z + dev_grid.guard[2] - shift;
  if (i < dev_grid.dims[0] - dev_grid.guard[0] + shift &&
      j < dev_grid.dims[1] - dev_grid.guard[1] + shift &&
      k < dev_grid.dims[2] - dev_grid.guard[2] + shift) {
    ijk = i + j * dev_grid.dims[0] +
          k * dev_grid.dims[0] * dev_grid.dims[1];
    // x:
    intex = interpolate(ex, ijk, Stagger(0b110), Stagger(0b110),
                        dev_grid.dims[0], dev_grid.dims[1]);
    intey = interpolate(ey, ijk, Stagger(0b101), Stagger(0b110),
                        dev_grid.dims[0], dev_grid.dims[1]);
    intez = interpolate(ez, ijk, Stagger(0b011), Stagger(0b110),
                        dev_grid.dims[0], dev_grid.dims[1]);
    emag = intex * intex + intey * intey + intez * intez + TINY;
    intbx = interpolate(bx, ijk, Stagger(0b001), Stagger(0b110),
                        dev_grid.dims[0], dev_grid.dims[1]);
    intby = interpolate(by, ijk, Stagger(0b010), Stagger(0b110),
                        dev_grid.dims[0], dev_grid.dims[1]);
    intbz = interpolate(bz, ijk, Stagger(0b100), Stagger(0b110),
                        dev_grid.dims[0], dev_grid.dims[1]);
    bmag = intbx * intbx + intby * intby + intbz * intbz + TINY;
    if (emag > bmag) {
      temp = sqrt(bmag / emag);
    } else {
      temp = 1.0;
    }
    ex[ijk] = temp * dex[ijk];

    // y:
    intex = interpolate(ex, ijk, Stagger(0b110), Stagger(0b101),
                        dev_grid.dims[0], dev_grid.dims[1]);
    intey = interpolate(ey, ijk, Stagger(0b101), Stagger(0b101),
                        dev_grid.dims[0], dev_grid.dims[1]);
    intez = interpolate(ez, ijk, Stagger(0b011), Stagger(0b101),
                        dev_grid.dims[0], dev_grid.dims[1]);
    emag = intex * intex + intey * intey + intez * intez + TINY;
    intbx = interpolate(bx, ijk, Stagger(0b001), Stagger(0b101),
                        dev_grid.dims[0], dev_grid.dims[1]);
    intby = interpolate(by, ijk, Stagger(0b010), Stagger(0b101),
                        dev_grid.dims[0], dev_grid.dims[1]);
    intbz = interpolate(bz, ijk, Stagger(0b100), Stagger(0b101),
                        dev_grid.dims[0], dev_grid.dims[1]);
    bmag = intbx * intbx + intby * intby + intbz * intbz + TINY;
    if (emag > bmag) {
      temp = sqrt(bmag / emag);
    } else {
      temp = 1.0;
    }
    ey[ijk] = temp * dey[ijk];

    // z:
    intex = interpolate(ex, ijk, Stagger(0b110), Stagger(0b011),
                        dev_grid.dims[0], dev_grid.dims[1]);
    intey = interpolate(ey, ijk, Stagger(0b101), Stagger(0b011),
                        dev_grid.dims[0], dev_grid.dims[1]);
    intez = interpolate(ez, ijk, Stagger(0b011), Stagger(0b011),
                        dev_grid.dims[0], dev_grid.dims[1]);
    emag = intex * intex + intey * intey + intez * intez + TINY;
    intbx = interpolate(bx, ijk, Stagger(0b001), Stagger(0b011),
                        dev_grid.dims[0], dev_grid.dims[1]);
    intby = interpolate(by, ijk, Stagger(0b010), Stagger(0b011),
                        dev_grid.dims[0], dev_grid.dims[1]);
    intbz = interpolate(bz, ijk, Stagger(0b100), Stagger(0b011),
                        dev_grid.dims[0], dev_grid.dims[1]);
    bmag = intbx * intbx + intby * intby + intbz * intbz + TINY;
    if (emag > bmag) {
      temp = sqrt(bmag / emag);
    } else {
      temp = 1.0;
    }
    ez[ijk] = temp * dez[ijk];
  }
}


field_solver::field_solver(sim_data &mydata, sim_environment& env) : m_data(mydata), m_env(env) {
  En = vector_field<Scalar>(m_data.env.grid());
  dE = vector_field<Scalar>(m_data.env.grid());
  En.copy_stagger(m_data.E);
  dE.copy_stagger(m_data.E);
  En.initialize();
  dE.initialize();

  Bn = vector_field<Scalar>(m_data.env.grid());
  dB = vector_field<Scalar>(m_data.env.grid());
  Bn.copy_stagger(m_data.B);
  dB.copy_stagger(m_data.B);
  Bn.initialize();
  dB.initialize();

  rho = multi_array<Scalar>(m_data.env.grid().extent());
  rho.assign_dev(0.0);

  blockGroupSize = dim3((m_data.env.grid().reduced_dim(0) + SHIFT_GHOST * 2 + blockSize.x - 1) / blockSize.x,
                        (m_data.env.grid().reduced_dim(1) + SHIFT_GHOST * 2 + blockSize.y - 1) / blockSize.y,
                        (m_data.env.grid().reduced_dim(2) + SHIFT_GHOST * 2 + blockSize.z - 1) / blockSize.z);
  std::cout << blockSize.x << ", " << blockSize.y << ", " << blockSize.z << std::endl;
  std::cout << blockGroupSize.x << ", " << blockGroupSize.y << ", " << blockGroupSize.z << std::endl;
}

field_solver::~field_solver() {}

void
field_solver::evolve_fields() {
  RANGE_PUSH("Compute", CLR_GREEN);
  copy_fields();

  // substep #1:
  rk_push();
  rk_update(1.0, 0.0, 1.0);
  check_eGTb();
  CudaSafeCall(hipDeviceSynchronize());
  RANGE_POP;
  m_env.send_guard_cells(m_data);

  // substep #2:
  RANGE_PUSH("Compute", CLR_GREEN);
  rk_push();
  rk_update(0.75, 0.25, 0.25);
  check_eGTb();
  CudaSafeCall(hipDeviceSynchronize());
  RANGE_POP;
  m_env.send_guard_cells(m_data);

  // substep #3:
  RANGE_PUSH("Compute", CLR_GREEN);
  rk_push();
  rk_update(1.0 / 3.0, 2.0 / 3.0, 2.0 / 3.0);
  clean_epar();
  check_eGTb();
  CudaSafeCall(hipDeviceSynchronize());
  RANGE_POP;

  m_env.send_guard_cells(m_data);
}

void
field_solver::copy_fields() {
  // `En = E, Bn = B`:
  En.copy_from(m_data.E);
  Bn.copy_from(m_data.B);
  dE.initialize();
  dB.initialize();
}

void
field_solver::rk_push() {
  // `rho = div E`
  kernel_compute_rho_thread<<<blockGroupSize, blockSize>>>(
      m_data.E.dev_ptr(0), m_data.E.dev_ptr(1), m_data.E.dev_ptr(2),
      rho.dev_ptr(), SHIFT_GHOST);
  // `dE = curl B - curl B0 - j, dB = -curl E`
  kernel_rk_push_thread<<<blockGroupSize, blockSize>>>(
      m_data.E.dev_ptr(0), m_data.E.dev_ptr(1), m_data.E.dev_ptr(2),
      m_data.B.dev_ptr(0), m_data.B.dev_ptr(1), m_data.B.dev_ptr(2),
      m_data.B0.dev_ptr(0), m_data.B0.dev_ptr(1), m_data.B0.dev_ptr(2),
      dE.dev_ptr(0), dE.dev_ptr(1), dE.dev_ptr(2), dB.dev_ptr(0),
      dB.dev_ptr(1), dB.dev_ptr(2), rho.dev_ptr(), SHIFT_GHOST);
}

void
field_solver::rk_update(Scalar rk_c1, Scalar rk_c2, Scalar rk_c3) {
  // `E = c1 En + c2 E + c3 dE, B = c1 Bn + c2 B + c3 dB`
  kernel_rk_update_thread<<<blockGroupSize, blockSize>>>(
      m_data.E.dev_ptr(0), m_data.E.dev_ptr(1), m_data.E.dev_ptr(2),
      m_data.B.dev_ptr(0), m_data.B.dev_ptr(1), m_data.B.dev_ptr(2),
      En.dev_ptr(0), En.dev_ptr(1), En.dev_ptr(2), Bn.dev_ptr(0),
      Bn.dev_ptr(1), Bn.dev_ptr(2), dE.dev_ptr(0), dE.dev_ptr(1),
      dE.dev_ptr(2), dB.dev_ptr(0), dB.dev_ptr(1), dB.dev_ptr(2), rk_c1,
      rk_c2, rk_c3, SHIFT_GHOST);
}

void
field_solver::clean_epar() {
  // clean `E || B`
  kernel_clean_epar_thread<<<blockGroupSize, blockSize>>>(
      m_data.E.dev_ptr(0), m_data.E.dev_ptr(1), m_data.E.dev_ptr(2),
      m_data.B.dev_ptr(0), m_data.B.dev_ptr(1), m_data.B.dev_ptr(2),
      dE.dev_ptr(0), dE.dev_ptr(1), dE.dev_ptr(2), SHIFT_GHOST);
}

void
field_solver::check_eGTb() {
  // renormalizing `E > B`
  kernel_check_eGTb_thread<<<blockGroupSize, blockSize>>>(
      dE.dev_ptr(0), dE.dev_ptr(1), dE.dev_ptr(2), m_data.E.dev_ptr(0),
      m_data.E.dev_ptr(1), m_data.E.dev_ptr(2), m_data.B.dev_ptr(0),
      m_data.B.dev_ptr(1), m_data.B.dev_ptr(2), SHIFT_GHOST);
}

}  // namespace Coffee
