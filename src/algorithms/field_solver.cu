#include "hip/hip_runtime.h"
#include "cuda/constant_mem.h"
#include "cuda/constant_mem_func.h"
#include "cuda/cuda_utility.h"
#include "field_solver.h"
#include "interpolation.h"
#include "utils/timer.h"
#include "utils/nvproftool.h"

#define BLOCK_SIZE_X 32
#define BLOCK_SIZE_Y 4
#define BLOCK_SIZE_Z 4

#define nghost 1

#define full_SIZE_X (BLOCK_SIZE_X + 2 * nghost)
#define full_SIZE_Y (BLOCK_SIZE_Y + 2 * nghost)
#define full_SIZE_Z (BLOCK_SIZE_Z + 2 * nghost)

#define SHIFT_GHOST 2

#define TINY 1e-7

namespace Coffee {

// static dim3 gridSize(8, 16, 16);
static dim3 blockSize(BLOCK_SIZE_X, BLOCK_SIZE_Y, BLOCK_SIZE_Z);

static dim3 blockGroupSize;

// __global__ void
// kernel_compute_rho(const Scalar *ex, const Scalar *ey, const Scalar *ez,
//                    Scalar *rho) {
//   size_t ijk, iM1jk, ijM1k, ijkM1;
//   for (int k = threadIdx.z + blockIdx.z * blockDim.z +
//                dev_grid.guard[2] - 1;
//        k < dev_grid.dims[2] - dev_grid.guard[2] + 1;
//        k += blockDim.z * gridDim.z) {
//     for (int j = threadIdx.y + blockIdx.y * blockDim.y +
//                  dev_grid.guard[1] - 1;
//          j < dev_grid.dims[1] - dev_grid.guard[1] + 1;
//          j += blockDim.y * gridDim.y) {
//       for (int i = threadIdx.x + blockIdx.x * blockDim.x +
//                    dev_grid.guard[0] - 1;
//            i < dev_grid.dims[0] - dev_grid.guard[0] + 1;
//            i += blockDim.x * gridDim.x) {
//         ijk = i + j * dev_grid.dims[0] +
//               k * dev_grid.dims[0] * dev_grid.dims[1];
//         iM1jk = ijk - 1;
//         ijM1k = ijk - dev_grid.dims[0];
//         ijkM1 = ijk - dev_grid.dims[0] * dev_grid.dims[1];
//         rho[ijk] = dev_grid.inv_delta[0] * (ex[ijk] - ex[iM1jk]) +
//                    dev_grid.inv_delta[1] * (ey[ijk] - ey[ijM1k]) +
//                    dev_grid.inv_delta[2] * (ez[ijk] - ez[ijkM1]);
//       }
//     }
//   }
// }

__global__ void
kernel_compute_rho_thread(const Scalar *ex, const Scalar *ey, const Scalar *ez,
                          Scalar *rho, int shift) {
  size_t ijk;
  int i = threadIdx.x + blockIdx.x * blockDim.x + dev_grid.guard[0] - shift;
  int j = threadIdx.y + blockIdx.y * blockDim.y + dev_grid.guard[1] - shift;
  int k = threadIdx.z + blockIdx.z * blockDim.z + dev_grid.guard[2] - shift;
  if (i < dev_grid.dims[0] - dev_grid.guard[0] + shift &&
      j < dev_grid.dims[1] - dev_grid.guard[1] + shift &&
      k < dev_grid.dims[2] - dev_grid.guard[2] + shift) {
    ijk = i + j * dev_grid.dims[0] +
          k * dev_grid.dims[0] * dev_grid.dims[1];
    rho[ijk] = dev_grid.inv_delta[0] * (ex[ijk] - ex[ijk - 1]) +
               dev_grid.inv_delta[1] * (ey[ijk] - ey[ijk - dev_grid.dims[0]]) +
               dev_grid.inv_delta[2] * (ez[ijk] - ez[ijk - dev_grid.dims[0] * dev_grid.dims[1]]);
  }
}

// __global__ void
// kernel_rk_push_shared(const Scalar *ex, const Scalar *ey, const Scalar *ez,
//                       const Scalar *bx, const Scalar *by, const Scalar *bz,
//                       const Scalar *bx0, const Scalar *by0, const Scalar *bz0,
//                       Scalar *dex, Scalar *dey, Scalar *dez, Scalar *dbx,
//                       Scalar *dby, Scalar *dbz, Scalar *rho) {
//   Scalar CCx = dev_params.dt * dev_grid.inv_delta[0];
//   Scalar CCy = dev_params.dt * dev_grid.inv_delta[1];
//   Scalar CCz = dev_params.dt * dev_grid.inv_delta[2];
//   Scalar intex, intey, intez, intbx, intby, intbz, intrho;
//   Scalar jx, jy, jz;
//
//   __shared__ Scalar
//       // sh_ex[(full_SIZE_Z) * (full_SIZE_Y) * (full_SIZE_X)];
//   sh_ex[full_SIZE_Z][full_SIZE_Y][full_SIZE_X];
//   __shared__ Scalar
//       // sh_ey[(full_SIZE_Z) * (full_SIZE_Y) * (full_SIZE_X)];
//   sh_ey[full_SIZE_Z][full_SIZE_Y][full_SIZE_X];
//   __shared__ Scalar
//       // sh_ez[(full_SIZE_Z) * (full_SIZE_Y) * (full_SIZE_X)];
//   sh_ez[full_SIZE_Z][full_SIZE_Y][full_SIZE_X];
//   __shared__ Scalar
//       // sh_bx[(full_SIZE_Z) * (full_SIZE_Y) * (full_SIZE_X)];
//   sh_bx[full_SIZE_Z][full_SIZE_Y][full_SIZE_X];
//   __shared__ Scalar
//       // sh_by[(full_SIZE_Z) * (full_SIZE_Y) * (full_SIZE_X)];
//   sh_by[full_SIZE_Z][full_SIZE_Y][full_SIZE_X];
//   __shared__ Scalar
//       // sh_bz[(full_SIZE_Z) * (full_SIZE_Y) * (full_SIZE_X)];
//   sh_bz[full_SIZE_Z][full_SIZE_Y][full_SIZE_X];
//   __shared__ Scalar
//       // sh_bx0[(full_SIZE_Z) * (full_SIZE_Y) * (full_SIZE_X)];
//   sh_bx0[full_SIZE_Z][full_SIZE_Y][full_SIZE_X];
//   __shared__ Scalar
//       // sh_by0[(full_SIZE_Z) * (full_SIZE_Y) * (full_SIZE_X)];
//   sh_by0[full_SIZE_Z][full_SIZE_Y][full_SIZE_X];
//   __shared__ Scalar
//       // sh_bz0[(full_SIZE_Z) * (full_SIZE_Y) * (full_SIZE_X)];
//   sh_bz0[full_SIZE_Z][full_SIZE_Y][full_SIZE_X];
//   __shared__ Scalar
//       // sh_rho[(full_SIZE_Z) * (full_SIZE_Y) * (full_SIZE_X)];
//   sh_rho[full_SIZE_Z][full_SIZE_Y][full_SIZE_X];
//
//   // populating __shared__ memory
//   size_t ijk, ijk_thr, iglob, jglob, kglob, i, j, k;
//   // size_t iP1jk_thr, iM1jk_thr, ijP1k_thr, ijM1k_thr, ijkP1_thr,
//   //     ijkM1_thr;
//   for (k = threadIdx.z; k < (full_SIZE_Z); k += blockDim.z) {
//     kglob = k + blockIdx.z * blockDim.z;
//     for (j = threadIdx.y; j < (full_SIZE_Y); j += blockDim.y) {
//       jglob = j + blockIdx.y * blockDim.y;
//       for (i = threadIdx.x; i < (full_SIZE_X); i += blockDim.x) {
//         iglob = i + blockIdx.x * blockDim.x;
//         if ((iglob < dev_grid.dims[0]) && (jglob < dev_grid.dims[1]) &&
//             (kglob < dev_grid.dims[2])) {
//           ijk = iglob + jglob * dev_grid.dims[0] +
//                 kglob * dev_grid.dims[0] * dev_grid.dims[1];
//           // ijk_thr =
//           //     i + j * (full_SIZE_X) + k * (full_SIZE_X) * (full_SIZE_Y);
//           // sh_ex[ijk_thr] = ex[ijk];
//           // sh_ey[ijk_thr] = ey[ijk];
//           // sh_ez[ijk_thr] = ez[ijk];
//           // sh_bx[ijk_thr] = bx[ijk];
//           // sh_by[ijk_thr] = by[ijk];
//           // sh_bz[ijk_thr] = bz[ijk];
//           // sh_bx0[ijk_thr] = bx0[ijk];
//           // sh_by0[ijk_thr] = by0[ijk];
//           // sh_bz0[ijk_thr] = bz0[ijk];
//           // sh_rho[ijk_thr] = rho[ijk];
//           sh_ex[k][j][i] = ex[ijk];
//           sh_ey[k][j][i] = ey[ijk];
//           sh_ez[k][j][i] = ez[ijk];
//           sh_bx[k][j][i] = bx[ijk];
//           sh_by[k][j][i] = by[ijk];
//           sh_bz[k][j][i] = bz[ijk];
//           sh_bx0[k][j][i] = bx0[ijk];
//           sh_by0[k][j][i] = by0[ijk];
//           sh_bz0[k][j][i] = bz0[ijk];
//           sh_rho[k][j][i] = rho[ijk];
//         }
//       }
//     }
//   }
//
//   // sync between threads
//   __syncthreads();
//   //
//
//   i = threadIdx.x + nghost;
//   j = threadIdx.y + nghost;
//   k = threadIdx.z + nghost;
//   iglob = i + blockIdx.x * blockDim.x;
//   jglob = j + blockIdx.y * blockDim.y;
//   kglob = k + blockIdx.z * blockDim.z;
//   if (((iglob < dev_grid.dims[0] - nghost) &&
//        (jglob < dev_grid.dims[1] - nghost) &&
//        (kglob < dev_grid.dims[2] - nghost))) {
//     // error
//     // asm("trap;");
//     ijk = iglob + jglob * dev_grid.dims[0] +
//           kglob * dev_grid.dims[0] * dev_grid.dims[1];
//     // ijk_thr = i + j * (full_SIZE_X) + k * (full_SIZE_X) * (full_SIZE_Y);
//
//     // iP1jk_thr = ijk_thr + 1;
//     // iM1jk_thr = ijk_thr - 1;
//     // ijP1k_thr = ijk_thr + (full_SIZE_X);
//     // ijM1k_thr = ijk_thr - (full_SIZE_X);
//     // ijkP1_thr = ijk_thr + (full_SIZE_X) * (full_SIZE_Y);
//     // ijkM1_thr = ijk_thr - (full_SIZE_X) * (full_SIZE_Y);
//
//     // // push B-field
//     // dbx[ijk] = CCx * (sh_ey[ijkP1_thr] - sh_ey[ijk_thr] -
//     //                   sh_ez[ijP1k_thr] + sh_ez[ijk_thr]);
//     // dby[ijk] = CCy * (sh_ez[iP1jk_thr] - sh_ez[ijk_thr] -
//     //                   sh_ez[ijkP1_thr] + sh_ex[ijk_thr]);
//     // dbz[ijk] = CCz * (sh_ex[ijP1k_thr] - sh_ex[ijk_thr] -
//     //                   sh_ez[iP1jk_thr] + sh_ey[ijk_thr]);
//     // // push E-field
//     // dex[ijk] = CCx * ((sh_by[ijkM1_thr] - sh_by[ijk_thr] -
//     //                    sh_bz[ijM1k_thr] + sh_bz[ijk_thr]) -
//     //                   (sh_by0[ijkM1_thr] - sh_bz0[ijk_thr] -
//     //                    sh_bz0[ijM1k_thr] + sh_bz0[ijk_thr]));
//     // dey[ijk] = CCy * ((sh_bz[iM1jk_thr] - sh_bz[ijk_thr] -
//     //                    sh_bx[ijkM1_thr] + sh_bx[ijk_thr]) -
//     //                   (sh_bz0[iM1jk_thr] - sh_bz0[ijk_thr] -
//     //                    sh_bx0[ijkM1_thr] + sh_bx0[ijk_thr]));
//     // dez[ijk] = CCz * ((sh_bx[ijM1k_thr] - sh_bx[ijk_thr] -
//     //                    sh_by[iM1jk_thr] + sh_by[ijk_thr]) -
//     //                   (sh_bx0[ijM1k_thr] - sh_bx0[ijk_thr] -
//     //                    sh_by0[iM1jk_thr] + sh_by0[ijk_thr]));
//     // push B-field
//     dbx[ijk] = CCx * (sh_ey[k + 1][j][i] - sh_ey[k][j][i] -
//                       sh_ez[k][j + 1][i] + sh_ez[k][j][i]);
//     dby[ijk] = CCy * (sh_ez[k][j][i + 1] - sh_ez[k][j][i] -
//                       sh_ex[k + 1][j][i] + sh_ex[k][j][i]);
//     dbz[ijk] = CCz * (sh_ex[k][j + 1][i] - sh_ex[k][j][i] -
//                       sh_ey[k][j][i + 1] + sh_ey[k][j][i]);
//     // push E-field
//     dex[ijk] = CCx * ((sh_by[k - 1][j][i] - sh_by[k][j][i] -
//                        sh_bz[k][j - 1][i] + sh_bz[k][j][i]) -
//                       (sh_by0[k - 1][j][i] - sh_by0[k][j][i] -
//                        sh_bz0[k][j - 1][i] + sh_bz0[k][j][i]));
//     dey[ijk] = CCy * ((sh_bz[k][j][i - 1] - sh_bz[k][j][i] -
//                        sh_bx[k - 1][j][i] + sh_bx[k][j][i]) -
//                       (sh_bz0[k][j][i - 1] - sh_bz0[k][j][i] -
//                        sh_bx0[k - 1][j][i] + sh_bx0[k][j][i]));
//     dez[ijk] = CCz * ((sh_bx[k][j - 1][i] - sh_bx[k][j][i] -
//                        sh_by[k][j][i - 1] + sh_by[k][j][i]) -
//                       (sh_bx0[k][j - 1][i] - sh_bx0[k][j][i] -
//                        sh_by0[k][j][i - 1] + sh_by0[k][j][i]));
//
//     //   `j_x` :
//     intrho = interpolate(sh_rho, i, j, k, Stagger(0b111), Stagger(0b011));
//     intex = interpolate(sh_ex, i, j, k, Stagger(0b011), Stagger(0b011));
//     intey = interpolate(sh_ey, i, j, k, Stagger(0b101), Stagger(0b011));
//     intez = interpolate(sh_ez, i, j, k, Stagger(0b110), Stagger(0b011));
//     intbx = interpolate(sh_bx, i, j, k, Stagger(0b001), Stagger(0b011));
//     intby = interpolate(sh_by, i, j, k, Stagger(0b010), Stagger(0b011));
//     intbz = interpolate(sh_bz, i, j, k, Stagger(0b001), Stagger(0b011));
//     jx = CCx * intrho * (intey * intbz - intby * intez) /
//          (intbx * intbx + intby * intby + intbz * intbz);
//     //   `j_y`:
//     intrho = interpolate(sh_rho, i, j, k, Stagger(0b111), Stagger(0b101));
//     intex = interpolate(sh_ex, i, j, k, Stagger(0b011), Stagger(0b101));
//     intey = interpolate(sh_ey, i, j, k, Stagger(0b101), Stagger(0b101));
//     intez = interpolate(sh_ez, i, j, k, Stagger(0b110), Stagger(0b101));
//     intbx = interpolate(sh_bx, i, j, k, Stagger(0b001), Stagger(0b101));
//     intby = interpolate(sh_by, i, j, k, Stagger(0b010), Stagger(0b101));
//     intbz = interpolate(sh_bz, i, j, k, Stagger(0b001), Stagger(0b101));
//     jy = CCy * intrho * (intez * intbx - intex * intbz) /
//          (intbx * intbx + intby * intby + intbz * intbz);
//     //   `j_z`:
//     intrho = interpolate(sh_rho, i, j, k, Stagger(0b111), Stagger(0b110));
//     intex = interpolate(sh_ex, i, j, k, Stagger(0b011), Stagger(0b110));
//     intey = interpolate(sh_ey, i, j, k, Stagger(0b101), Stagger(0b110));
//     intez = interpolate(sh_ez, i, j, k, Stagger(0b110), Stagger(0b110));
//     intbx = interpolate(sh_bx, i, j, k, Stagger(0b001), Stagger(0b110));
//     intby = interpolate(sh_by, i, j, k, Stagger(0b010), Stagger(0b110));
//     intbz = interpolate(sh_bz, i, j, k, Stagger(0b001), Stagger(0b110));
//     jz = CCz * intrho * (intex * intby - intbx * intey) /
//          (intbx * intbx + intby * intby + intbz * intbz);
//
//     dex[ijk] -= jx;
//     dey[ijk] -= jy;
//     dez[ijk] -= jz;
//   }
// }

__global__ void
kernel_rk_push_thread(const Scalar *ex, const Scalar *ey, const Scalar *ez,
                      const Scalar *bx, const Scalar *by, const Scalar *bz,
                      const Scalar *bx0, const Scalar *by0,
                      const Scalar *bz0, Scalar *dex, Scalar *dey,
                      Scalar *dez, Scalar *dbx, Scalar *dby, Scalar *dbz,
                      Scalar *rho, int shift) {
  Scalar CCx = dev_params.dt * dev_grid.inv_delta[0];
  Scalar CCy = dev_params.dt * dev_grid.inv_delta[1];
  Scalar CCz = dev_params.dt * dev_grid.inv_delta[2];
  Scalar intex, intey, intez, intbx, intby, intbz, intrho;
  Scalar jx, jy, jz;
  size_t ijk, iP1jk, iM1jk, ijP1k, ijM1k, ijkP1, ijkM1;

  int i = threadIdx.x + blockIdx.x * blockDim.x + dev_grid.guard[0] - shift;
  int j = threadIdx.y + blockIdx.y * blockDim.y + dev_grid.guard[1] - shift;
  int k = threadIdx.z + blockIdx.z * blockDim.z + dev_grid.guard[2] - shift;
  if (i < dev_grid.dims[0] - dev_grid.guard[0] + shift &&
      j < dev_grid.dims[1] - dev_grid.guard[1] + shift &&
      k < dev_grid.dims[2] - dev_grid.guard[2] + shift) {
    ijk = i + j * dev_grid.dims[0] +
          k * dev_grid.dims[0] * dev_grid.dims[1];
    iP1jk = ijk + 1;
    iM1jk = ijk - 1;
    ijP1k = ijk + dev_grid.dims[0];
    ijM1k = ijk - dev_grid.dims[0];
    ijkP1 = ijk + dev_grid.dims[0] * dev_grid.dims[1];
    ijkM1 = ijk - dev_grid.dims[0] * dev_grid.dims[1];
    // push B-field
    dbx[ijk] = CCz * (ey[ijkP1] - ey[ijk]) - CCy * (ez[ijP1k] - ez[ijk]);
    dby[ijk] = CCx * (ez[iP1jk] - ez[ijk]) - CCz * (ex[ijkP1] - ex[ijk]);
    dbz[ijk] = CCy * (ex[ijP1k] - ex[ijk]) - CCx * (ey[iP1jk] - ey[ijk]);
    // push E-field
    dex[ijk] = (CCz * (by[ijkM1] - by[ijk]) - CCy * (bz[ijM1k] - bz[ijk])) -
                      (CCz * (by0[ijkM1] - bz0[ijk]) - CCy * (bz0[ijM1k] - bz0[ijk]));
    dey[ijk] = (CCx * (bz[iM1jk] - bz[ijk]) - CCz * (bx[ijkM1] - bx[ijk])) -
                      (CCx * (bz0[iM1jk] - bz0[ijk]) - CCz * (bx0[ijkM1] - bx0[ijk]));
    dez[ijk] = (CCy * (bx[ijM1k] - bx[ijk]) - CCx * (by[iM1jk] - by[ijk])) -
                      (CCy * (bx0[ijM1k] - bx0[ijk]) - CCx * (by0[iM1jk] - by0[ijk]));
    // if (i == 10 && j == 10 && k == 10)
      // printf("%d, %d, %d\n", dev_grid.dims[0], dev_grid.dims[1], dev_grid.dims[2]);
      // printf("%f, %f, %f\n", dex[ijk], dey[ijk], dez[ijk]);
      // printf("%lu, %lu, %lu\n", ijkM1, ijM1k, iM1jk);
    // computing currents
    //   `j_x`:
    intrho = interpolate(rho, ijk, Stagger(0b111), Stagger(0b110),
                         dev_grid.dims[0], dev_grid.dims[1]);
    intex = interpolate(ex, ijk, Stagger(0b110), Stagger(0b110),
                       dev_grid.dims[0], dev_grid.dims[1]);
    intey = interpolate(ey, ijk, Stagger(0b101), Stagger(0b110),
                       dev_grid.dims[0], dev_grid.dims[1]);
    intez = interpolate(ez, ijk, Stagger(0b011), Stagger(0b110),
                       dev_grid.dims[0], dev_grid.dims[1]);
    intbx = interpolate(bx, ijk, Stagger(0b001), Stagger(0b110),
                       dev_grid.dims[0], dev_grid.dims[1]);
    intby = interpolate(by, ijk, Stagger(0b010), Stagger(0b110),
                       dev_grid.dims[0], dev_grid.dims[1]);
    intbz = interpolate(bz, ijk, Stagger(0b100), Stagger(0b110),
                       dev_grid.dims[0], dev_grid.dims[1]);
    jx = dev_params.dt * intrho * (intey * intbz - intby * intez) /
         (intbx * intbx + intby * intby + intbz * intbz + TINY);
    //   `j_y`:
    intrho = interpolate(rho, ijk, Stagger(0b111), Stagger(0b101),
                         dev_grid.dims[0], dev_grid.dims[1]);
    intex = interpolate(ex, ijk, Stagger(0b110), Stagger(0b101),
                       dev_grid.dims[0], dev_grid.dims[1]);
    intey = interpolate(ey, ijk, Stagger(0b101), Stagger(0b101),
                       dev_grid.dims[0], dev_grid.dims[1]);
    intez = interpolate(ez, ijk, Stagger(0b011), Stagger(0b101),
                       dev_grid.dims[0], dev_grid.dims[1]);
    intbx = interpolate(bx, ijk, Stagger(0b001), Stagger(0b101),
                       dev_grid.dims[0], dev_grid.dims[1]);
    intby = interpolate(by, ijk, Stagger(0b010), Stagger(0b101),
                       dev_grid.dims[0], dev_grid.dims[1]);
    intbz = interpolate(bz, ijk, Stagger(0b100), Stagger(0b101),
                       dev_grid.dims[0], dev_grid.dims[1]);
    jy = dev_params.dt * intrho * (intez * intbx - intex * intbz) /
         (intbx * intbx + intby * intby + intbz * intbz + TINY);
    //   `j_z`:
    intrho = interpolate(rho, ijk, Stagger(0b111), Stagger(0b011),
                         dev_grid.dims[0], dev_grid.dims[1]);
    intex = interpolate(ex, ijk, Stagger(0b110), Stagger(0b011),
                       dev_grid.dims[0], dev_grid.dims[1]);
    intey = interpolate(ey, ijk, Stagger(0b101), Stagger(0b011),
                       dev_grid.dims[0], dev_grid.dims[1]);
    intez = interpolate(ez, ijk, Stagger(0b011), Stagger(0b011),
                       dev_grid.dims[0], dev_grid.dims[1]);
    intbx = interpolate(bx, ijk, Stagger(0b001), Stagger(0b011),
                       dev_grid.dims[0], dev_grid.dims[1]);
    intby = interpolate(by, ijk, Stagger(0b010), Stagger(0b011),
                       dev_grid.dims[0], dev_grid.dims[1]);
    intbz = interpolate(bz, ijk, Stagger(0b100), Stagger(0b011),
                       dev_grid.dims[0], dev_grid.dims[1]);
    jz = dev_params.dt * intrho * (intex * intby - intbx * intey) /
         (intbx * intbx + intby * intby + intbz * intbz + TINY);

    dex[ijk] -= jx;
    dey[ijk] -= jy;
    dez[ijk] -= jz;
  }
}

// __global__ void
// kernel_rk_update(Scalar *ex, Scalar *ey, Scalar *ez, Scalar *bx,
//                  Scalar *by, Scalar *bz, const Scalar *enx,
//                  const Scalar *eny, const Scalar *enz,
//                  const Scalar *bnx, const Scalar *bny,
//                  const Scalar *bnz, Scalar *dex, Scalar *dey,
//                  Scalar *dez, const Scalar *dbx, const Scalar *dby,
//                  const Scalar *dbz, Scalar rk_c1, Scalar rk_c2,
//                  Scalar rk_c3) {
//   size_t ijk;
//   for (int k =
//            threadIdx.z + blockIdx.z * blockDim.z + dev_grid.guard[2];
//        k < dev_grid.dims[2] - dev_grid.guard[2];
//        k += blockDim.z * gridDim.z) {
//     for (int j =
//              threadIdx.y + blockIdx.y * blockDim.y + dev_grid.guard[1];
//          j < dev_grid.dims[1] - dev_grid.guard[1];
//          j += blockDim.y * gridDim.y) {
//       for (int i = threadIdx.x + blockIdx.x * blockDim.x +
//                    dev_grid.guard[0];
//            i < dev_grid.dims[0] - dev_grid.guard[0];
//            i += blockDim.x * gridDim.x) {
//         ijk = i + j * dev_grid.dims[0] +
//               k * dev_grid.dims[0] * dev_grid.dims[1];
//         // update E-field
//         ex[ijk] = rk_c1 * enx[ijk] + rk_c2 * ex[ijk] + rk_c3 * dex[ijk];
//         ey[ijk] = rk_c1 * eny[ijk] + rk_c2 * ey[ijk] + rk_c3 * dey[ijk];
//         ez[ijk] = rk_c1 * enz[ijk] + rk_c2 * ez[ijk] + rk_c3 * dez[ijk];
//         dex[ijk] = ex[ijk];
//         dey[ijk] = ey[ijk];
//         dez[ijk] = ez[ijk];
//         // update B-field
//         bx[ijk] = rk_c1 * bnx[ijk] + rk_c2 * bx[ijk] + rk_c3 * dbx[ijk];
//         by[ijk] = rk_c1 * bny[ijk] + rk_c2 * by[ijk] + rk_c3 * dby[ijk];
//         bz[ijk] = rk_c1 * bnz[ijk] + rk_c2 * bz[ijk] + rk_c3 * dbz[ijk];
//       }
//     }
//   }
// }

__global__ void
kernel_rk_update_thread(Scalar *ex, Scalar *ey, Scalar *ez, Scalar *bx,
                        Scalar *by, Scalar *bz, const Scalar *enx,
                        const Scalar *eny, const Scalar *enz,
                        const Scalar *bnx, const Scalar *bny,
                        const Scalar *bnz, Scalar *dex, Scalar *dey,
                        Scalar *dez, const Scalar *dbx, const Scalar *dby,
                        const Scalar *dbz, Scalar rk_c1, Scalar rk_c2,
                        Scalar rk_c3, int shift) {
  size_t ijk;
  int i = threadIdx.x + blockIdx.x * blockDim.x + dev_grid.guard[0] - shift;
  int j = threadIdx.y + blockIdx.y * blockDim.y + dev_grid.guard[1] - shift;
  int k = threadIdx.z + blockIdx.z * blockDim.z + dev_grid.guard[2] - shift;
  if (i < dev_grid.dims[0] - dev_grid.guard[0] + shift &&
      j < dev_grid.dims[1] - dev_grid.guard[1] + shift &&
      k < dev_grid.dims[2] - dev_grid.guard[2] + shift) {
    ijk = i + j * dev_grid.dims[0] +
          k * dev_grid.dims[0] * dev_grid.dims[1];
    // update E-field
    ex[ijk] = rk_c1 * enx[ijk] + rk_c2 * ex[ijk] + rk_c3 * dex[ijk];
    ey[ijk] = rk_c1 * eny[ijk] + rk_c2 * ey[ijk] + rk_c3 * dey[ijk];
    ez[ijk] = rk_c1 * enz[ijk] + rk_c2 * ez[ijk] + rk_c3 * dez[ijk];
    dex[ijk] = ex[ijk];
    dey[ijk] = ey[ijk];
    dez[ijk] = ez[ijk];
    // update B-field
    bx[ijk] = rk_c1 * bnx[ijk] + rk_c2 * bx[ijk] + rk_c3 * dbx[ijk];
    by[ijk] = rk_c1 * bny[ijk] + rk_c2 * by[ijk] + rk_c3 * dby[ijk];
    bz[ijk] = rk_c1 * bnz[ijk] + rk_c2 * bz[ijk] + rk_c3 * dbz[ijk];
  }
}

// __global__ void
// kernel_clean_epar(const Scalar *ex, const Scalar *ey, const Scalar *ez,
//                   const Scalar *bx, const Scalar *by, const Scalar *bz,
//                   Scalar *dex, Scalar *dey, Scalar *dez) {
//   Scalar intex, intey, intez, intbx, intby, intbz;
//   size_t ijk;
//   for (int k =
//            threadIdx.z + blockIdx.z * blockDim.z + dev_grid.guard[2];
//        k < dev_grid.dims[2] - dev_grid.guard[2];
//        k += blockDim.z * gridDim.z) {
//     for (int j =
//              threadIdx.y + blockIdx.y * blockDim.y + dev_grid.guard[1];
//          j < dev_grid.dims[1] - dev_grid.guard[1];
//          j += blockDim.y * gridDim.y) {
//       for (int i = threadIdx.x + blockIdx.x * blockDim.x +
//                    dev_grid.guard[0];
//            i < dev_grid.dims[0] - dev_grid.guard[0];
//            i += blockDim.x * gridDim.x) {
//         ijk = i + j * dev_grid.dims[0] +
//               k * dev_grid.dims[0] * dev_grid.dims[1];
//         // x:
//         intex = interpolate(ex, ijk, Stagger(0b011), Stagger(0b011),
//                             dev_grid.dims[0], dev_grid.dims[1]);
//         intey = interpolate(ey, ijk, Stagger(0b101), Stagger(0b011),
//                             dev_grid.dims[0], dev_grid.dims[1]);
//         intez = interpolate(ez, ijk, Stagger(0b110), Stagger(0b011),
//                             dev_grid.dims[0], dev_grid.dims[1]);
//         intbx = interpolate(bx, ijk, Stagger(0b001), Stagger(0b011),
//                             dev_grid.dims[0], dev_grid.dims[1]);
//         intby = interpolate(by, ijk, Stagger(0b010), Stagger(0b011),
//                             dev_grid.dims[0], dev_grid.dims[1]);
//         intbz = interpolate(bz, ijk, Stagger(0b001), Stagger(0b011),
//                             dev_grid.dims[0], dev_grid.dims[1]);
//         dex[ijk] = ex[ijk] -
//                    (intex * intbx + intey * intby + intez * intbz) *
//                        intbx /
//                        (intbx * intbx + intby * intby + intbz * intbz);
//
//         // y:
//         intex = interpolate(ex, ijk, Stagger(0b011), Stagger(0b101),
//                             dev_grid.dims[0], dev_grid.dims[1]);
//         intey = interpolate(ey, ijk, Stagger(0b101), Stagger(0b101),
//                             dev_grid.dims[0], dev_grid.dims[1]);
//         intez = interpolate(ez, ijk, Stagger(0b110), Stagger(0b101),
//                             dev_grid.dims[0], dev_grid.dims[1]);
//         intbx = interpolate(bx, ijk, Stagger(0b001), Stagger(0b101),
//                             dev_grid.dims[0], dev_grid.dims[1]);
//         intby = interpolate(by, ijk, Stagger(0b010), Stagger(0b101),
//                             dev_grid.dims[0], dev_grid.dims[1]);
//         intbz = interpolate(bz, ijk, Stagger(0b001), Stagger(0b101),
//                             dev_grid.dims[0], dev_grid.dims[1]);
//         dey[ijk] = ey[ijk] -
//                    (intex * intbx + intey * intby + intez * intbz) *
//                        intby /
//                        (intbx * intbx + intby * intby + intbz * intbz);
//
//         // z:
//         intex = interpolate(ex, ijk, Stagger(0b011), Stagger(0b110),
//                             dev_grid.dims[0], dev_grid.dims[1]);
//         intey = interpolate(ey, ijk, Stagger(0b101), Stagger(0b110),
//                             dev_grid.dims[0], dev_grid.dims[1]);
//         intez = interpolate(ez, ijk, Stagger(0b110), Stagger(0b110),
//                             dev_grid.dims[0], dev_grid.dims[1]);
//         intbx = interpolate(bx, ijk, Stagger(0b001), Stagger(0b110),
//                             dev_grid.dims[0], dev_grid.dims[1]);
//         intby = interpolate(by, ijk, Stagger(0b010), Stagger(0b110),
//                             dev_grid.dims[0], dev_grid.dims[1]);
//         intbz = interpolate(bz, ijk, Stagger(0b001), Stagger(0b110),
//                             dev_grid.dims[0], dev_grid.dims[1]);
//         dez[ijk] = ez[ijk] -
//                    (intex * intbx + intey * intby + intez * intbz) *
//                        intbz /
//                        (intbx * intbx + intby * intby + intbz * intbz);
//       }
//     }
//   }
// }

__global__ void
kernel_clean_epar_thread(const Scalar *ex, const Scalar *ey, const Scalar *ez,
                         const Scalar *bx, const Scalar *by, const Scalar *bz,
                         Scalar *dex, Scalar *dey, Scalar *dez, int shift) {
  Scalar intex, intey, intez, intbx, intby, intbz;
  size_t ijk;
  int i = threadIdx.x + blockIdx.x * blockDim.x + dev_grid.guard[0] - shift;
  int j = threadIdx.y + blockIdx.y * blockDim.y + dev_grid.guard[1] - shift;
  int k = threadIdx.z + blockIdx.z * blockDim.z + dev_grid.guard[2] - shift;
  if (i < dev_grid.dims[0] - dev_grid.guard[0] + shift &&
      j < dev_grid.dims[1] - dev_grid.guard[1] + shift &&
      k < dev_grid.dims[2] - dev_grid.guard[2] + shift) {
    ijk = i + j * dev_grid.dims[0] +
          k * dev_grid.dims[0] * dev_grid.dims[1];
    // x:
    intex = interpolate(ex, ijk, Stagger(0b110), Stagger(0b110),
                        dev_grid.dims[0], dev_grid.dims[1]);
    intey = interpolate(ey, ijk, Stagger(0b101), Stagger(0b110),
                        dev_grid.dims[0], dev_grid.dims[1]);
    intez = interpolate(ez, ijk, Stagger(0b011), Stagger(0b110),
                        dev_grid.dims[0], dev_grid.dims[1]);
    intbx = interpolate(bx, ijk, Stagger(0b001), Stagger(0b110),
                        dev_grid.dims[0], dev_grid.dims[1]);
    intby = interpolate(by, ijk, Stagger(0b010), Stagger(0b110),
                        dev_grid.dims[0], dev_grid.dims[1]);
    intbz = interpolate(bz, ijk, Stagger(0b100), Stagger(0b110),
                        dev_grid.dims[0], dev_grid.dims[1]);
    dex[ijk] = ex[ijk] -
               (intex * intbx + intey * intby + intez * intbz) *
                   intbx /
                   (intbx * intbx + intby * intby + intbz * intbz + TINY);

    // y:
    intex = interpolate(ex, ijk, Stagger(0b110), Stagger(0b101),
                        dev_grid.dims[0], dev_grid.dims[1]);
    intey = interpolate(ey, ijk, Stagger(0b101), Stagger(0b101),
                        dev_grid.dims[0], dev_grid.dims[1]);
    intez = interpolate(ez, ijk, Stagger(0b011), Stagger(0b101),
                        dev_grid.dims[0], dev_grid.dims[1]);
    intbx = interpolate(bx, ijk, Stagger(0b001), Stagger(0b101),
                        dev_grid.dims[0], dev_grid.dims[1]);
    intby = interpolate(by, ijk, Stagger(0b010), Stagger(0b101),
                        dev_grid.dims[0], dev_grid.dims[1]);
    intbz = interpolate(bz, ijk, Stagger(0b100), Stagger(0b101),
                        dev_grid.dims[0], dev_grid.dims[1]);
    dey[ijk] = ey[ijk] -
               (intex * intbx + intey * intby + intez * intbz) *
                   intby /
                   (intbx * intbx + intby * intby + intbz * intbz + TINY);

    // z:
    intex = interpolate(ex, ijk, Stagger(0b110), Stagger(0b011),
                        dev_grid.dims[0], dev_grid.dims[1]);
    intey = interpolate(ey, ijk, Stagger(0b101), Stagger(0b011),
                        dev_grid.dims[0], dev_grid.dims[1]);
    intez = interpolate(ez, ijk, Stagger(0b011), Stagger(0b011),
                        dev_grid.dims[0], dev_grid.dims[1]);
    intbx = interpolate(bx, ijk, Stagger(0b001), Stagger(0b011),
                        dev_grid.dims[0], dev_grid.dims[1]);
    intby = interpolate(by, ijk, Stagger(0b010), Stagger(0b011),
                        dev_grid.dims[0], dev_grid.dims[1]);
    intbz = interpolate(bz, ijk, Stagger(0b100), Stagger(0b011),
                        dev_grid.dims[0], dev_grid.dims[1]);
    dez[ijk] = ez[ijk] -
               (intex * intbx + intey * intby + intez * intbz) *
                   intbz /
                   (intbx * intbx + intby * intby + intbz * intbz + TINY);
  }
}

// __global__ void
// kernel_check_eGTb(const Scalar *dex, const Scalar *dey,
//                   const Scalar *dez, Scalar *ex, Scalar *ey, Scalar *ez,
//                   const Scalar *bx, const Scalar *by,
//                   const Scalar *bz) {
//   Scalar intex, intey, intez, intbx, intby, intbz, emag, bmag, temp;
//   size_t ijk;
//   for (int k =
//            threadIdx.z + blockIdx.z * blockDim.z + dev_grid.guard[2];
//        k < dev_grid.dims[2] - dev_grid.guard[2];
//        k += blockDim.z * gridDim.z) {
//     for (int j =
//              threadIdx.y + blockIdx.y * blockDim.y + dev_grid.guard[1];
//          j < dev_grid.dims[1] - dev_grid.guard[1];
//          j += blockDim.y * gridDim.y) {
//       for (int i = threadIdx.x + blockIdx.x * blockDim.x +
//                    dev_grid.guard[0];
//            i < dev_grid.dims[0] - dev_grid.guard[0];
//            i += blockDim.x * gridDim.x) {
//         ijk = i + j * dev_grid.dims[0] +
//               k * dev_grid.dims[0] * dev_grid.dims[1];
//         // x:
//         intex = interpolate(ex, ijk, Stagger(0b011), Stagger(0b011),
//                             dev_grid.dims[0], dev_grid.dims[1]);
//         intey = interpolate(ey, ijk, Stagger(0b101), Stagger(0b011),
//                             dev_grid.dims[0], dev_grid.dims[1]);
//         intez = interpolate(ez, ijk, Stagger(0b110), Stagger(0b011),
//                             dev_grid.dims[0], dev_grid.dims[1]);
//         emag = intex * intex + intey * intey + intez * intez;
//         intbx = interpolate(bx, ijk, Stagger(0b001), Stagger(0b011),
//                             dev_grid.dims[0], dev_grid.dims[1]);
//         intby = interpolate(by, ijk, Stagger(0b010), Stagger(0b011),
//                             dev_grid.dims[0], dev_grid.dims[1]);
//         intbz = interpolate(bz, ijk, Stagger(0b001), Stagger(0b011),
//                             dev_grid.dims[0], dev_grid.dims[1]);
//         bmag = intbx * intbx + intby * intby + intbz * intbz;
//         if (emag > bmag) {
//           temp = sqrt(bmag / emag);
//         } else {
//           temp = 1.0;
//         }
//         ex[ijk] = temp * dex[ijk];
//
//         // y:
//         intex = interpolate(ex, ijk, Stagger(0b011), Stagger(0b101),
//                             dev_grid.dims[0], dev_grid.dims[1]);
//         intey = interpolate(ey, ijk, Stagger(0b101), Stagger(0b101),
//                             dev_grid.dims[0], dev_grid.dims[1]);
//         intez = interpolate(ez, ijk, Stagger(0b110), Stagger(0b101),
//                             dev_grid.dims[0], dev_grid.dims[1]);
//         emag = intex * intex + intey * intey + intez * intez;
//         intbx = interpolate(bx, ijk, Stagger(0b001), Stagger(0b101),
//                             dev_grid.dims[0], dev_grid.dims[1]);
//         intby = interpolate(by, ijk, Stagger(0b010), Stagger(0b101),
//                             dev_grid.dims[0], dev_grid.dims[1]);
//         intbz = interpolate(bz, ijk, Stagger(0b001), Stagger(0b101),
//                             dev_grid.dims[0], dev_grid.dims[1]);
//         bmag = intbx * intbx + intby * intby + intbz * intbz;
//         if (emag > bmag) {
//           temp = sqrt(bmag / emag);
//         } else {
//           temp = 1.0;
//         }
//         ey[ijk] = temp * dey[ijk];
//
//         // z:
//         intex = interpolate(ex, ijk, Stagger(0b011), Stagger(0b110),
//                             dev_grid.dims[0], dev_grid.dims[1]);
//         intey = interpolate(ey, ijk, Stagger(0b101), Stagger(0b110),
//                             dev_grid.dims[0], dev_grid.dims[1]);
//         intez = interpolate(ez, ijk, Stagger(0b110), Stagger(0b110),
//                             dev_grid.dims[0], dev_grid.dims[1]);
//         emag = intex * intex + intey * intey + intez * intez;
//         intbx = interpolate(bx, ijk, Stagger(0b001), Stagger(0b110),
//                             dev_grid.dims[0], dev_grid.dims[1]);
//         intby = interpolate(by, ijk, Stagger(0b010), Stagger(0b110),
//                             dev_grid.dims[0], dev_grid.dims[1]);
//         intbz = interpolate(bz, ijk, Stagger(0b001), Stagger(0b110),
//                             dev_grid.dims[0], dev_grid.dims[1]);
//         bmag = intbx * intbx + intby * intby + intbz * intbz;
//         if (emag > bmag) {
//           temp = sqrt(bmag / emag);
//         } else {
//           temp = 1.0;
//         }
//         ez[ijk] = temp * dez[ijk];
//       }
//     }
//   }
// }

__global__ void
kernel_check_eGTb_thread(const Scalar *dex, const Scalar *dey,
                         const Scalar *dez, Scalar *ex, Scalar *ey, Scalar *ez,
                         const Scalar *bx, const Scalar *by,
                         const Scalar *bz, int shift) {
  Scalar intex, intey, intez, intbx, intby, intbz, emag, bmag, temp;
  size_t ijk;
  int i = threadIdx.x + blockIdx.x * blockDim.x + dev_grid.guard[0] - shift;
  int j = threadIdx.y + blockIdx.y * blockDim.y + dev_grid.guard[1] - shift;
  int k = threadIdx.z + blockIdx.z * blockDim.z + dev_grid.guard[2] - shift;
  if (i < dev_grid.dims[0] - dev_grid.guard[0] + shift &&
      j < dev_grid.dims[1] - dev_grid.guard[1] + shift &&
      k < dev_grid.dims[2] - dev_grid.guard[2] + shift) {
    ijk = i + j * dev_grid.dims[0] +
          k * dev_grid.dims[0] * dev_grid.dims[1];
    // x:
    intex = interpolate(ex, ijk, Stagger(0b110), Stagger(0b110),
                        dev_grid.dims[0], dev_grid.dims[1]);
    intey = interpolate(ey, ijk, Stagger(0b101), Stagger(0b110),
                        dev_grid.dims[0], dev_grid.dims[1]);
    intez = interpolate(ez, ijk, Stagger(0b011), Stagger(0b110),
                        dev_grid.dims[0], dev_grid.dims[1]);
    emag = intex * intex + intey * intey + intez * intez + TINY;
    intbx = interpolate(bx, ijk, Stagger(0b001), Stagger(0b110),
                        dev_grid.dims[0], dev_grid.dims[1]);
    intby = interpolate(by, ijk, Stagger(0b010), Stagger(0b110),
                        dev_grid.dims[0], dev_grid.dims[1]);
    intbz = interpolate(bz, ijk, Stagger(0b100), Stagger(0b110),
                        dev_grid.dims[0], dev_grid.dims[1]);
    bmag = intbx * intbx + intby * intby + intbz * intbz + TINY;
    if (emag > bmag) {
      temp = sqrt(bmag / emag);
    } else {
      temp = 1.0;
    }
    ex[ijk] = temp * dex[ijk];

    // y:
    intex = interpolate(ex, ijk, Stagger(0b110), Stagger(0b101),
                        dev_grid.dims[0], dev_grid.dims[1]);
    intey = interpolate(ey, ijk, Stagger(0b101), Stagger(0b101),
                        dev_grid.dims[0], dev_grid.dims[1]);
    intez = interpolate(ez, ijk, Stagger(0b011), Stagger(0b101),
                        dev_grid.dims[0], dev_grid.dims[1]);
    emag = intex * intex + intey * intey + intez * intez + TINY;
    intbx = interpolate(bx, ijk, Stagger(0b001), Stagger(0b101),
                        dev_grid.dims[0], dev_grid.dims[1]);
    intby = interpolate(by, ijk, Stagger(0b010), Stagger(0b101),
                        dev_grid.dims[0], dev_grid.dims[1]);
    intbz = interpolate(bz, ijk, Stagger(0b100), Stagger(0b101),
                        dev_grid.dims[0], dev_grid.dims[1]);
    bmag = intbx * intbx + intby * intby + intbz * intbz + TINY;
    if (emag > bmag) {
      temp = sqrt(bmag / emag);
    } else {
      temp = 1.0;
    }
    ey[ijk] = temp * dey[ijk];

    // z:
    intex = interpolate(ex, ijk, Stagger(0b110), Stagger(0b011),
                        dev_grid.dims[0], dev_grid.dims[1]);
    intey = interpolate(ey, ijk, Stagger(0b101), Stagger(0b011),
                        dev_grid.dims[0], dev_grid.dims[1]);
    intez = interpolate(ez, ijk, Stagger(0b011), Stagger(0b011),
                        dev_grid.dims[0], dev_grid.dims[1]);
    emag = intex * intex + intey * intey + intez * intez + TINY;
    intbx = interpolate(bx, ijk, Stagger(0b001), Stagger(0b011),
                        dev_grid.dims[0], dev_grid.dims[1]);
    intby = interpolate(by, ijk, Stagger(0b010), Stagger(0b011),
                        dev_grid.dims[0], dev_grid.dims[1]);
    intbz = interpolate(bz, ijk, Stagger(0b100), Stagger(0b011),
                        dev_grid.dims[0], dev_grid.dims[1]);
    bmag = intbx * intbx + intby * intby + intbz * intbz + TINY;
    if (emag > bmag) {
      temp = sqrt(bmag / emag);
    } else {
      temp = 1.0;
    }
    ez[ijk] = temp * dez[ijk];
  }
}


field_solver::field_solver(sim_data &mydata, sim_environment& env) : m_data(mydata), m_env(env) {
  En = vector_field<Scalar>(m_data.env.grid());
  dE = vector_field<Scalar>(m_data.env.grid());
  En.copy_stagger(m_data.E);
  dE.copy_stagger(m_data.E);
  En.initialize();
  dE.initialize();

  Bn = vector_field<Scalar>(m_data.env.grid());
  dB = vector_field<Scalar>(m_data.env.grid());
  Bn.copy_stagger(m_data.B);
  dB.copy_stagger(m_data.B);
  Bn.initialize();
  dB.initialize();

  rho = multi_array<Scalar>(m_data.env.grid().extent());
  rho.assign_dev(0.0);

  blockGroupSize = dim3((m_data.env.grid().reduced_dim(0) + SHIFT_GHOST * 2 + blockSize.x - 1) / blockSize.x,
                        (m_data.env.grid().reduced_dim(1) + SHIFT_GHOST * 2 + blockSize.y - 1) / blockSize.y,
                        (m_data.env.grid().reduced_dim(2) + SHIFT_GHOST * 2 + blockSize.z - 1) / blockSize.z);
  std::cout << blockSize.x << ", " << blockSize.y << ", " << blockSize.z << std::endl;
  std::cout << blockGroupSize.x << ", " << blockGroupSize.y << ", " << blockGroupSize.z << std::endl;
}

field_solver::~field_solver() {}

void
field_solver::evolve_fields() {
  RANGE_PUSH("Compute", CLR_GREEN);
  copy_fields();

  // substep #1:
  rk_push();
  rk_update(1.0, 0.0, 1.0);
  check_eGTb();
  CudaSafeCall(hipDeviceSynchronize());
  RANGE_POP;
  m_env.send_guard_cells(m_data);

  // substep #2:
  RANGE_PUSH("Compute", CLR_GREEN);
  rk_push();
  rk_update(0.75, 0.25, 0.25);
  check_eGTb();
  CudaSafeCall(hipDeviceSynchronize());
  RANGE_POP;
  m_env.send_guard_cells(m_data);

  // substep #3:
  RANGE_PUSH("Compute", CLR_GREEN);
  rk_push();
  rk_update(1.0 / 3.0, 2.0 / 3.0, 2.0 / 3.0);
  clean_epar();
  check_eGTb();
  CudaSafeCall(hipDeviceSynchronize());
  RANGE_POP;

  m_env.send_guard_cells(m_data);
}

void
field_solver::copy_fields() {
  // `En = E, Bn = B`:
  En.copy_from(m_data.E);
  Bn.copy_from(m_data.B);
  dE.initialize();
  dB.initialize();
}

void
field_solver::rk_push() {
  // `rho = div E`
  // kernel_compute_rho<<<gridSize, blockSize>>>(
  kernel_compute_rho_thread<<<blockGroupSize, blockSize>>>(
      m_data.E.dev_ptr(0), m_data.E.dev_ptr(1), m_data.E.dev_ptr(2),
      rho.dev_ptr(), SHIFT_GHOST);
  CudaCheckError();
  // `dE = curl B - curl B0 - j, dB = -curl E`
  // kernel_rk_push<<<g, blockSize>>>(
  kernel_rk_push_thread<<<blockGroupSize, blockSize>>>(
      m_data.E.dev_ptr(0), m_data.E.dev_ptr(1), m_data.E.dev_ptr(2),
      m_data.B.dev_ptr(0), m_data.B.dev_ptr(1), m_data.B.dev_ptr(2),
      m_data.B0.dev_ptr(0), m_data.B0.dev_ptr(1), m_data.B0.dev_ptr(2),
      dE.dev_ptr(0), dE.dev_ptr(1), dE.dev_ptr(2), dB.dev_ptr(0),
      dB.dev_ptr(1), dB.dev_ptr(2), rho.dev_ptr(), SHIFT_GHOST);
  CudaCheckError();
}

void
field_solver::rk_update(Scalar rk_c1, Scalar rk_c2, Scalar rk_c3) {
  // `E = c1 En + c2 E + c3 dE, B = c1 Bn + c2 B + c3 dB`
  // kernel_rk_update<<<dim3(8, 16, 16), dim3(64, 4, 4)>>>(
  kernel_rk_update_thread<<<blockGroupSize, blockSize>>>(
      m_data.E.dev_ptr(0), m_data.E.dev_ptr(1), m_data.E.dev_ptr(2),
      m_data.B.dev_ptr(0), m_data.B.dev_ptr(1), m_data.B.dev_ptr(2),
      En.dev_ptr(0), En.dev_ptr(1), En.dev_ptr(2), Bn.dev_ptr(0),
      Bn.dev_ptr(1), Bn.dev_ptr(2), dE.dev_ptr(0), dE.dev_ptr(1),
      dE.dev_ptr(2), dB.dev_ptr(0), dB.dev_ptr(1), dB.dev_ptr(2), rk_c1,
      rk_c2, rk_c3, SHIFT_GHOST);
  CudaCheckError();
}

void
field_solver::clean_epar() {
  // clean `E || B`
  // kernel_clean_epar<<<gridSize, blockSize>>>(
  kernel_clean_epar_thread<<<blockGroupSize, blockSize>>>(
      m_data.E.dev_ptr(0), m_data.E.dev_ptr(1), m_data.E.dev_ptr(2),
      m_data.B.dev_ptr(0), m_data.B.dev_ptr(1), m_data.B.dev_ptr(2),
      dE.dev_ptr(0), dE.dev_ptr(1), dE.dev_ptr(2), SHIFT_GHOST);
  CudaCheckError();
}

void
field_solver::check_eGTb() {
  // renormalizing `E > B`
  // kernel_check_eGTb<<<dim3(8, 16, 16), dim3(32, 4, 4)>>>(
  kernel_check_eGTb_thread<<<blockGroupSize, blockSize>>>(
      dE.dev_ptr(0), dE.dev_ptr(1), dE.dev_ptr(2), m_data.E.dev_ptr(0),
      m_data.E.dev_ptr(1), m_data.E.dev_ptr(2), m_data.B.dev_ptr(0),
      m_data.B.dev_ptr(1), m_data.B.dev_ptr(2), SHIFT_GHOST);
  CudaCheckError();
}

}  // namespace Coffee
