#include "hip/hip_runtime.h"
#include "cuda/constant_mem.h"
#include "cuda/constant_mem_func.h"
#include "cuda/cuda_utility.h"
#include "field_solver.h"
#include "interpolation.h"
#include "boundary.h"
#include "pulsar.h"
#include "utils/nvproftool.h"
#include "utils/timer.h"

#define BLOCK_SIZE_X 32
#define BLOCK_SIZE_Y 4
#define BLOCK_SIZE_Z 4

namespace Coffee {

// static dim3 gridSize(8, 16, 16);
static dim3 blockSize(BLOCK_SIZE_X, BLOCK_SIZE_Y, BLOCK_SIZE_Z);

static dim3 blockGroupSize;

__global__ void
kernel_compute_rho_thread(const Scalar *ex, const Scalar *ey,
                          const Scalar *ez, Scalar *rho, int shift) {
  size_t ijk;
  int i =
      threadIdx.x + blockIdx.x * blockDim.x + dev_grid.guard[0] - shift;
  int j =
      threadIdx.y + blockIdx.y * blockDim.y + dev_grid.guard[1] - shift;
  int k =
      threadIdx.z + blockIdx.z * blockDim.z + dev_grid.guard[2] - shift;
  if (i < dev_grid.dims[0] - dev_grid.guard[0] + shift &&
      j < dev_grid.dims[1] - dev_grid.guard[1] + shift &&
      k < dev_grid.dims[2] - dev_grid.guard[2] + shift) {
    ijk = i + j * dev_grid.dims[0] +
          k * dev_grid.dims[0] * dev_grid.dims[1];
    rho[ijk] =
        dev_grid.inv_delta[0] * (ex[ijk] - ex[ijk - 1]) +
        dev_grid.inv_delta[1] * (ey[ijk] - ey[ijk - dev_grid.dims[0]]) +
        dev_grid.inv_delta[2] *
            (ez[ijk] - ez[ijk - dev_grid.dims[0] * dev_grid.dims[1]]);
  }
}

__global__ void
kernel_rk_push_thread(const Scalar *ex, const Scalar *ey,
                      const Scalar *ez, const Scalar *bx,
                      const Scalar *by, const Scalar *bz,
                      const Scalar *bx0, const Scalar *by0,
                      const Scalar *bz0, Scalar *dex, Scalar *dey,
                      Scalar *dez, Scalar *dbx, Scalar *dby,
                      Scalar *dbz, Scalar *rho, int shift) {
  Scalar CCx = dev_params.dt * dev_grid.inv_delta[0];
  Scalar CCy = dev_params.dt * dev_grid.inv_delta[1];
  Scalar CCz = dev_params.dt * dev_grid.inv_delta[2];
  Scalar intex, intey, intez, intbx, intby, intbz, intrho;
  Scalar jx, jy, jz;
  size_t ijk, iP1jk, iM1jk, ijP1k, ijM1k, ijkP1, ijkM1;

  int i =
      threadIdx.x + blockIdx.x * blockDim.x + dev_grid.guard[0] - shift;
  int j =
      threadIdx.y + blockIdx.y * blockDim.y + dev_grid.guard[1] - shift;
  int k =
      threadIdx.z + blockIdx.z * blockDim.z + dev_grid.guard[2] - shift;
  if (i < dev_grid.dims[0] - dev_grid.guard[0] + shift &&
      j < dev_grid.dims[1] - dev_grid.guard[1] + shift &&
      k < dev_grid.dims[2] - dev_grid.guard[2] + shift) {
    ijk = i + j * dev_grid.dims[0] +
          k * dev_grid.dims[0] * dev_grid.dims[1];
    iP1jk = ijk + 1;
    iM1jk = ijk - 1;
    ijP1k = ijk + dev_grid.dims[0];
    ijM1k = ijk - dev_grid.dims[0];
    ijkP1 = ijk + dev_grid.dims[0] * dev_grid.dims[1];
    ijkM1 = ijk - dev_grid.dims[0] * dev_grid.dims[1];
    // push B-field
    dbx[ijk] =
        CCz * (ey[ijkP1] - ey[ijk]) - CCy * (ez[ijP1k] - ez[ijk]);
    dby[ijk] =
        CCx * (ez[iP1jk] - ez[ijk]) - CCz * (ex[ijkP1] - ex[ijk]);
    dbz[ijk] =
        CCy * (ex[ijP1k] - ex[ijk]) - CCx * (ey[iP1jk] - ey[ijk]);
    // push E-field
    dex[ijk] =
        (CCz * (by[ijkM1] - by[ijk]) - CCy * (bz[ijM1k] - bz[ijk])) -
        (CCz * (by0[ijkM1] - by0[ijk]) - CCy * (bz0[ijM1k] - bz0[ijk]));
    dey[ijk] =
        (CCx * (bz[iM1jk] - bz[ijk]) - CCz * (bx[ijkM1] - bx[ijk])) -
        (CCx * (bz0[iM1jk] - bz0[ijk]) - CCz * (bx0[ijkM1] - bx0[ijk]));
    dez[ijk] =
        (CCy * (bx[ijM1k] - bx[ijk]) - CCx * (by[iM1jk] - by[ijk])) -
        (CCy * (bx0[ijM1k] - bx0[ijk]) - CCx * (by0[iM1jk] - by0[ijk]));

    // computing currents
    //   `j_x`:
    intrho = interpolate(rho, ijk, Stagger(0b111), Stagger(0b110),
                         dev_grid.dims[0], dev_grid.dims[1]);
    intex = interpolate(ex, ijk, Stagger(0b110), Stagger(0b110),
                        dev_grid.dims[0], dev_grid.dims[1]);
    intey = interpolate(ey, ijk, Stagger(0b101), Stagger(0b110),
                        dev_grid.dims[0], dev_grid.dims[1]);
    intez = interpolate(ez, ijk, Stagger(0b011), Stagger(0b110),
                        dev_grid.dims[0], dev_grid.dims[1]);
    intbx = interpolate(bx, ijk, Stagger(0b001), Stagger(0b110),
                        dev_grid.dims[0], dev_grid.dims[1]);
    intby = interpolate(by, ijk, Stagger(0b010), Stagger(0b110),
                        dev_grid.dims[0], dev_grid.dims[1]);
    intbz = interpolate(bz, ijk, Stagger(0b100), Stagger(0b110),
                        dev_grid.dims[0], dev_grid.dims[1]);
    jx = dev_params.dt * intrho * (intey * intbz - intby * intez) /
         (intbx * intbx + intby * intby + intbz * intbz + TINY);
    //   `j_y`:
    intrho = interpolate(rho, ijk, Stagger(0b111), Stagger(0b101),
                         dev_grid.dims[0], dev_grid.dims[1]);
    intex = interpolate(ex, ijk, Stagger(0b110), Stagger(0b101),
                        dev_grid.dims[0], dev_grid.dims[1]);
    intey = interpolate(ey, ijk, Stagger(0b101), Stagger(0b101),
                        dev_grid.dims[0], dev_grid.dims[1]);
    intez = interpolate(ez, ijk, Stagger(0b011), Stagger(0b101),
                        dev_grid.dims[0], dev_grid.dims[1]);
    intbx = interpolate(bx, ijk, Stagger(0b001), Stagger(0b101),
                        dev_grid.dims[0], dev_grid.dims[1]);
    intby = interpolate(by, ijk, Stagger(0b010), Stagger(0b101),
                        dev_grid.dims[0], dev_grid.dims[1]);
    intbz = interpolate(bz, ijk, Stagger(0b100), Stagger(0b101),
                        dev_grid.dims[0], dev_grid.dims[1]);
    jy = dev_params.dt * intrho * (intez * intbx - intex * intbz) /
         (intbx * intbx + intby * intby + intbz * intbz + TINY);
    //   `j_z`:
    intrho = interpolate(rho, ijk, Stagger(0b111), Stagger(0b011),
                         dev_grid.dims[0], dev_grid.dims[1]);
    intex = interpolate(ex, ijk, Stagger(0b110), Stagger(0b011),
                        dev_grid.dims[0], dev_grid.dims[1]);
    intey = interpolate(ey, ijk, Stagger(0b101), Stagger(0b011),
                        dev_grid.dims[0], dev_grid.dims[1]);
    intez = interpolate(ez, ijk, Stagger(0b011), Stagger(0b011),
                        dev_grid.dims[0], dev_grid.dims[1]);
    intbx = interpolate(bx, ijk, Stagger(0b001), Stagger(0b011),
                        dev_grid.dims[0], dev_grid.dims[1]);
    intby = interpolate(by, ijk, Stagger(0b010), Stagger(0b011),
                        dev_grid.dims[0], dev_grid.dims[1]);
    intbz = interpolate(bz, ijk, Stagger(0b100), Stagger(0b011),
                        dev_grid.dims[0], dev_grid.dims[1]);
    jz = dev_params.dt * intrho * (intex * intby - intbx * intey) /
         (intbx * intbx + intby * intby + intbz * intbz + TINY);

    dex[ijk] -= jx;
    dey[ijk] -= jy;
    dez[ijk] -= jz;
  }
}

__global__ void
kernel_rk_update_thread(Scalar *ex, Scalar *ey, Scalar *ez, Scalar *bx,
                        Scalar *by, Scalar *bz, const Scalar *enx,
                        const Scalar *eny, const Scalar *enz,
                        const Scalar *bnx, const Scalar *bny,
                        const Scalar *bnz, Scalar *dex, Scalar *dey,
                        Scalar *dez, const Scalar *dbx,
                        const Scalar *dby, const Scalar *dbz,
                        Scalar rk_c1, Scalar rk_c2, Scalar rk_c3,
                        int shift) {
  size_t ijk;
  int i =
      threadIdx.x + blockIdx.x * blockDim.x + dev_grid.guard[0] - shift;
  int j =
      threadIdx.y + blockIdx.y * blockDim.y + dev_grid.guard[1] - shift;
  int k =
      threadIdx.z + blockIdx.z * blockDim.z + dev_grid.guard[2] - shift;
  if (i < dev_grid.dims[0] - dev_grid.guard[0] + shift &&
      j < dev_grid.dims[1] - dev_grid.guard[1] + shift &&
      k < dev_grid.dims[2] - dev_grid.guard[2] + shift) {
    ijk = i + j * dev_grid.dims[0] +
          k * dev_grid.dims[0] * dev_grid.dims[1];
    // update E-field
    ex[ijk] = rk_c1 * enx[ijk] + rk_c2 * ex[ijk] + rk_c3 * dex[ijk];
    ey[ijk] = rk_c1 * eny[ijk] + rk_c2 * ey[ijk] + rk_c3 * dey[ijk];
    ez[ijk] = rk_c1 * enz[ijk] + rk_c2 * ez[ijk] + rk_c3 * dez[ijk];
    dex[ijk] = ex[ijk];
    dey[ijk] = ey[ijk];
    dez[ijk] = ez[ijk];
    // update B-field
    bx[ijk] = rk_c1 * bnx[ijk] + rk_c2 * bx[ijk] + rk_c3 * dbx[ijk];
    by[ijk] = rk_c1 * bny[ijk] + rk_c2 * by[ijk] + rk_c3 * dby[ijk];
    bz[ijk] = rk_c1 * bnz[ijk] + rk_c2 * bz[ijk] + rk_c3 * dbz[ijk];
  }
}

__global__ void
kernel_clean_epar_thread(const Scalar *ex, const Scalar *ey,
                         const Scalar *ez, const Scalar *bx,
                         const Scalar *by, const Scalar *bz,
                         Scalar *dex, Scalar *dey, Scalar *dez,
                         int shift) {
  Scalar intex, intey, intez, intbx, intby, intbz;
  size_t ijk;
  int i =
      threadIdx.x + blockIdx.x * blockDim.x + dev_grid.guard[0] - shift;
  int j =
      threadIdx.y + blockIdx.y * blockDim.y + dev_grid.guard[1] - shift;
  int k =
      threadIdx.z + blockIdx.z * blockDim.z + dev_grid.guard[2] - shift;
  if (i < dev_grid.dims[0] - dev_grid.guard[0] + shift &&
      j < dev_grid.dims[1] - dev_grid.guard[1] + shift &&
      k < dev_grid.dims[2] - dev_grid.guard[2] + shift) {
    ijk = i + j * dev_grid.dims[0] +
          k * dev_grid.dims[0] * dev_grid.dims[1];
    // x:
    intex = interpolate(ex, ijk, Stagger(0b110), Stagger(0b110),
                        dev_grid.dims[0], dev_grid.dims[1]);
    intey = interpolate(ey, ijk, Stagger(0b101), Stagger(0b110),
                        dev_grid.dims[0], dev_grid.dims[1]);
    intez = interpolate(ez, ijk, Stagger(0b011), Stagger(0b110),
                        dev_grid.dims[0], dev_grid.dims[1]);
    intbx = interpolate(bx, ijk, Stagger(0b001), Stagger(0b110),
                        dev_grid.dims[0], dev_grid.dims[1]);
    intby = interpolate(by, ijk, Stagger(0b010), Stagger(0b110),
                        dev_grid.dims[0], dev_grid.dims[1]);
    intbz = interpolate(bz, ijk, Stagger(0b100), Stagger(0b110),
                        dev_grid.dims[0], dev_grid.dims[1]);
    dex[ijk] =
        ex[ijk] -
        (intex * intbx + intey * intby + intez * intbz) * intbx /
            (intbx * intbx + intby * intby + intbz * intbz + TINY);

    // y:
    intex = interpolate(ex, ijk, Stagger(0b110), Stagger(0b101),
                        dev_grid.dims[0], dev_grid.dims[1]);
    intey = interpolate(ey, ijk, Stagger(0b101), Stagger(0b101),
                        dev_grid.dims[0], dev_grid.dims[1]);
    intez = interpolate(ez, ijk, Stagger(0b011), Stagger(0b101),
                        dev_grid.dims[0], dev_grid.dims[1]);
    intbx = interpolate(bx, ijk, Stagger(0b001), Stagger(0b101),
                        dev_grid.dims[0], dev_grid.dims[1]);
    intby = interpolate(by, ijk, Stagger(0b010), Stagger(0b101),
                        dev_grid.dims[0], dev_grid.dims[1]);
    intbz = interpolate(bz, ijk, Stagger(0b100), Stagger(0b101),
                        dev_grid.dims[0], dev_grid.dims[1]);
    dey[ijk] =
        ey[ijk] -
        (intex * intbx + intey * intby + intez * intbz) * intby /
            (intbx * intbx + intby * intby + intbz * intbz + TINY);

    // z:
    intex = interpolate(ex, ijk, Stagger(0b110), Stagger(0b011),
                        dev_grid.dims[0], dev_grid.dims[1]);
    intey = interpolate(ey, ijk, Stagger(0b101), Stagger(0b011),
                        dev_grid.dims[0], dev_grid.dims[1]);
    intez = interpolate(ez, ijk, Stagger(0b011), Stagger(0b011),
                        dev_grid.dims[0], dev_grid.dims[1]);
    intbx = interpolate(bx, ijk, Stagger(0b001), Stagger(0b011),
                        dev_grid.dims[0], dev_grid.dims[1]);
    intby = interpolate(by, ijk, Stagger(0b010), Stagger(0b011),
                        dev_grid.dims[0], dev_grid.dims[1]);
    intbz = interpolate(bz, ijk, Stagger(0b100), Stagger(0b011),
                        dev_grid.dims[0], dev_grid.dims[1]);
    dez[ijk] =
        ez[ijk] -
        (intex * intbx + intey * intby + intez * intbz) * intbz /
            (intbx * intbx + intby * intby + intbz * intbz + TINY);
  }
}

__global__ void
kernel_check_eGTb_thread(const Scalar *dex, const Scalar *dey,
                         const Scalar *dez, Scalar *ex, Scalar *ey,
                         Scalar *ez, const Scalar *bx, const Scalar *by,
                         const Scalar *bz, int shift) {
  Scalar intex, intey, intez, intbx, intby, intbz, emag, bmag, temp;
  size_t ijk;
  int i =
      threadIdx.x + blockIdx.x * blockDim.x + dev_grid.guard[0] - shift;
  int j =
      threadIdx.y + blockIdx.y * blockDim.y + dev_grid.guard[1] - shift;
  int k =
      threadIdx.z + blockIdx.z * blockDim.z + dev_grid.guard[2] - shift;
  if (i < dev_grid.dims[0] - dev_grid.guard[0] + shift &&
      j < dev_grid.dims[1] - dev_grid.guard[1] + shift &&
      k < dev_grid.dims[2] - dev_grid.guard[2] + shift) {
    ijk = i + j * dev_grid.dims[0] +
          k * dev_grid.dims[0] * dev_grid.dims[1];
    // x:
    intex = interpolate(ex, ijk, Stagger(0b110), Stagger(0b110),
                        dev_grid.dims[0], dev_grid.dims[1]);
    intey = interpolate(ey, ijk, Stagger(0b101), Stagger(0b110),
                        dev_grid.dims[0], dev_grid.dims[1]);
    intez = interpolate(ez, ijk, Stagger(0b011), Stagger(0b110),
                        dev_grid.dims[0], dev_grid.dims[1]);
    emag = intex * intex + intey * intey + intez * intez + TINY;
    intbx = interpolate(bx, ijk, Stagger(0b001), Stagger(0b110),
                        dev_grid.dims[0], dev_grid.dims[1]);
    intby = interpolate(by, ijk, Stagger(0b010), Stagger(0b110),
                        dev_grid.dims[0], dev_grid.dims[1]);
    intbz = interpolate(bz, ijk, Stagger(0b100), Stagger(0b110),
                        dev_grid.dims[0], dev_grid.dims[1]);
    bmag = intbx * intbx + intby * intby + intbz * intbz + TINY;
    if (emag > bmag) {
      temp = sqrt(bmag / emag);
    } else {
      temp = 1.0;
    }
    ex[ijk] = temp * dex[ijk];

    // y:
    intex = interpolate(ex, ijk, Stagger(0b110), Stagger(0b101),
                        dev_grid.dims[0], dev_grid.dims[1]);
    intey = interpolate(ey, ijk, Stagger(0b101), Stagger(0b101),
                        dev_grid.dims[0], dev_grid.dims[1]);
    intez = interpolate(ez, ijk, Stagger(0b011), Stagger(0b101),
                        dev_grid.dims[0], dev_grid.dims[1]);
    emag = intex * intex + intey * intey + intez * intez + TINY;
    intbx = interpolate(bx, ijk, Stagger(0b001), Stagger(0b101),
                        dev_grid.dims[0], dev_grid.dims[1]);
    intby = interpolate(by, ijk, Stagger(0b010), Stagger(0b101),
                        dev_grid.dims[0], dev_grid.dims[1]);
    intbz = interpolate(bz, ijk, Stagger(0b100), Stagger(0b101),
                        dev_grid.dims[0], dev_grid.dims[1]);
    bmag = intbx * intbx + intby * intby + intbz * intbz + TINY;
    if (emag > bmag) {
      temp = sqrt(bmag / emag);
    } else {
      temp = 1.0;
    }
    ey[ijk] = temp * dey[ijk];

    // z:
    intex = interpolate(ex, ijk, Stagger(0b110), Stagger(0b011),
                        dev_grid.dims[0], dev_grid.dims[1]);
    intey = interpolate(ey, ijk, Stagger(0b101), Stagger(0b011),
                        dev_grid.dims[0], dev_grid.dims[1]);
    intez = interpolate(ez, ijk, Stagger(0b011), Stagger(0b011),
                        dev_grid.dims[0], dev_grid.dims[1]);
    emag = intex * intex + intey * intey + intez * intez + TINY;
    intbx = interpolate(bx, ijk, Stagger(0b001), Stagger(0b011),
                        dev_grid.dims[0], dev_grid.dims[1]);
    intby = interpolate(by, ijk, Stagger(0b010), Stagger(0b011),
                        dev_grid.dims[0], dev_grid.dims[1]);
    intbz = interpolate(bz, ijk, Stagger(0b100), Stagger(0b011),
                        dev_grid.dims[0], dev_grid.dims[1]);
    bmag = intbx * intbx + intby * intby + intbz * intbz + TINY;
    if (emag > bmag) {
      temp = sqrt(bmag / emag);
    } else {
      temp = 1.0;
    }
    ez[ijk] = temp * dez[ijk];
  }
}

__global__ void
kernel_boundary_pulsar_B_thread(Scalar *Bx, Scalar *By, Scalar *Bz,
                                Scalar *Bxnew, Scalar *Bynew,
                                Scalar *Bznew, Scalar t, int shift) {
  size_t ijk;
  Scalar x, y, z, r2, r, s;
  Scalar bxn, byn, bzn;
  Scalar intbx, intby, intbz;
  int i =
      threadIdx.x + blockIdx.x * blockDim.x + dev_grid.guard[0] - shift;
  int j =
      threadIdx.y + blockIdx.y * blockDim.y + dev_grid.guard[1] - shift;
  int k =
      threadIdx.z + blockIdx.z * blockDim.z + dev_grid.guard[2] - shift;
  if (i < dev_grid.dims[0] - dev_grid.guard[0] + shift &&
      j < dev_grid.dims[1] - dev_grid.guard[1] + shift &&
      k < dev_grid.dims[2] - dev_grid.guard[2] + shift) {
    ijk = i + j * dev_grid.dims[0] +
          k * dev_grid.dims[0] * dev_grid.dims[1];
    Scalar x0 = dev_grid.pos(0, i, 1);
    Scalar y0 = dev_grid.pos(1, j, 1);
    Scalar z0 = dev_grid.pos(2, k, 1);
    Scalar rl = 2.0 * dev_params.radius;
    // Smoothing scale
    Scalar scaleEpar = 1.0 * dev_grid.delta[0];
    Scalar scaleEperp = 0.5 * dev_grid.delta[0];
    Scalar scaleBperp = scaleEpar;
    Scalar scaleBpar = scaleBperp;
    Scalar d1 = 4 * dev_grid.delta[0];
    Scalar d0 = 0;
    Scalar phase = dev_params.omega * t;

    if (x0 * x0 + y0 * y0 + z0 * z0 < rl * rl) {
      // Set bx
      x = dev_grid.pos(0, i, 1);
      y = dev_grid.pos(1, j, 0);
      z = dev_grid.pos(2, k, 0);
      r2 = x * x + y * y + z * z;
      r = std::sqrt(r2);
      bxn = dev_params.b0 *
            quadru_dipole(
                x, y, z, dev_params.p1, dev_params.p2, dev_params.p3,
                dev_params.q11, dev_params.q12, dev_params.q13,
                dev_params.q22, dev_params.q23, dev_params.q_offset_x,
                dev_params.q_offset_y, dev_params.q_offset_z, phase, 0);
      byn = dev_params.b0 *
            quadru_dipole(
                x, y, z, dev_params.p1, dev_params.p2, dev_params.p3,
                dev_params.q11, dev_params.q12, dev_params.q13,
                dev_params.q22, dev_params.q23, dev_params.q_offset_x,
                dev_params.q_offset_y, dev_params.q_offset_z, phase, 1);
      bzn = dev_params.b0 *
            quadru_dipole(
                x, y, z, dev_params.p1, dev_params.p2, dev_params.p3,
                dev_params.q11, dev_params.q12, dev_params.q13,
                dev_params.q22, dev_params.q23, dev_params.q_offset_x,
                dev_params.q_offset_y, dev_params.q_offset_z, phase, 2);
      s = shape(r, dev_params.radius - d1, scaleBperp);
      intbx = Bx[ijk];
      intby = interpolate(By, ijk, Stagger(0b010), Stagger(0b001),
                          dev_grid.dims[0], dev_grid.dims[1]);
      intbz = interpolate(Bz, ijk, Stagger(0b100), Stagger(0b001),
                          dev_grid.dims[0], dev_grid.dims[1]);
      Bxnew[ijk] =
          (bxn * x + byn * y + bzn * z) * x / r2 * s +
          (intbx * x + intby * y + intbz * z) * x / r2 * (1 - s);

      s = shape(r, dev_params.radius - d1, scaleBpar);
      Bxnew[ijk] +=
          (bxn - (bxn * x + byn * y + bzn * z) * x / r2) * s +
          (intbx - (intbx * x + intby * y + intbz * z) * x / r2) *
              (1 - s);
      // Set by
      x = dev_grid.pos(0, i, 0);
      y = dev_grid.pos(1, j, 1);
      z = dev_grid.pos(2, k, 0);
      r2 = x * x + y * y + z * z;
      r = std::sqrt(r2);
      bxn = dev_params.b0 *
            quadru_dipole(
                x, y, z, dev_params.p1, dev_params.p2, dev_params.p3,
                dev_params.q11, dev_params.q12, dev_params.q13,
                dev_params.q22, dev_params.q23, dev_params.q_offset_x,
                dev_params.q_offset_y, dev_params.q_offset_z, phase, 0);
      byn = dev_params.b0 *
            quadru_dipole(
                x, y, z, dev_params.p1, dev_params.p2, dev_params.p3,
                dev_params.q11, dev_params.q12, dev_params.q13,
                dev_params.q22, dev_params.q23, dev_params.q_offset_x,
                dev_params.q_offset_y, dev_params.q_offset_z, phase, 1);
      bzn = dev_params.b0 *
            quadru_dipole(
                x, y, z, dev_params.p1, dev_params.p2, dev_params.p3,
                dev_params.q11, dev_params.q12, dev_params.q13,
                dev_params.q22, dev_params.q23, dev_params.q_offset_x,
                dev_params.q_offset_y, dev_params.q_offset_z, phase, 2);
      s = shape(r, dev_params.radius - d1, scaleBperp);
      intbx = interpolate(Bx, ijk, Stagger(0b001), Stagger(0b010),
                          dev_grid.dims[0], dev_grid.dims[1]);
      intby = By[ijk];
      intbz = interpolate(Bz, ijk, Stagger(0b100), Stagger(0b010),
                          dev_grid.dims[0], dev_grid.dims[1]);
      Bynew[ijk] =
          (bxn * x + byn * y + bzn * z) * y / r2 * s +
          (intbx * x + intby * y + intbz * z) * y / r2 * (1 - s);

      s = shape(r, dev_params.radius - d1, scaleBpar);
      Bynew[ijk] +=
          (byn - (bxn * x + byn * y + bzn * z) * y / r2) * s +
          (By[ijk] - (intbx * x + intby * y + intbz * z) * y / r2) *
              (1 - s);
      // Set bz
      x = dev_grid.pos(0, i, 0);
      y = dev_grid.pos(1, j, 0);
      z = dev_grid.pos(2, k, 1);
      r2 = x * x + y * y + z * z;
      r = std::sqrt(r2);
      bxn = dev_params.b0 *
            quadru_dipole(
                x, y, z, dev_params.p1, dev_params.p2, dev_params.p3,
                dev_params.q11, dev_params.q12, dev_params.q13,
                dev_params.q22, dev_params.q23, dev_params.q_offset_x,
                dev_params.q_offset_y, dev_params.q_offset_z, phase, 0);
      byn = dev_params.b0 *
            quadru_dipole(
                x, y, z, dev_params.p1, dev_params.p2, dev_params.p3,
                dev_params.q11, dev_params.q12, dev_params.q13,
                dev_params.q22, dev_params.q23, dev_params.q_offset_x,
                dev_params.q_offset_y, dev_params.q_offset_z, phase, 1);
      bzn = dev_params.b0 *
            quadru_dipole(
                x, y, z, dev_params.p1, dev_params.p2, dev_params.p3,
                dev_params.q11, dev_params.q12, dev_params.q13,
                dev_params.q22, dev_params.q23, dev_params.q_offset_x,
                dev_params.q_offset_y, dev_params.q_offset_z, phase, 2);
      s = shape(r, dev_params.radius - d1, scaleBperp);
      intbx = interpolate(Bx, ijk, Stagger(0b001), Stagger(0b100),
                          dev_grid.dims[0], dev_grid.dims[1]);
      intby = interpolate(By, ijk, Stagger(0b010), Stagger(0b100),
                          dev_grid.dims[0], dev_grid.dims[1]);
      intbz = Bz[ijk];
      Bznew[ijk] =
          (bxn * x + byn * y + bzn * z) * z / r2 * s +
          (intbx * x + intby * y + intbz * z) * z / r2 * (1 - s);

      s = shape(r, dev_params.radius - d1, scaleBpar);
      Bznew[ijk] +=
          (bzn - (bxn * x + byn * y + bzn * z) * z / r2) * s +
          (Bz[ijk] - (intbx * x + intby * y + intbz * z) * z / r2) *
              (1 - s);
    } else {
      Bxnew[ijk] = Bx[ijk];
      Bynew[ijk] = By[ijk];
      Bznew[ijk] = Bz[ijk];
    }
  }
}

__global__ void
kernel_boundary_pulsar_E_thread(Scalar *Ex, Scalar *Ey, Scalar *Ez,
                                Scalar *Bx, Scalar *By, Scalar *Bz,
                                Scalar *Exnew, Scalar *Eynew,
                                Scalar *Eznew, Scalar t, int shift) {
  size_t ijk;
  Scalar x, y, z, r2, r, s;
  Scalar exn, eyn, ezn, vx, vy;
  Scalar intex, intey, intez, intbx, intby, intbz;
  int i =
      threadIdx.x + blockIdx.x * blockDim.x + dev_grid.guard[0] - shift;
  int j =
      threadIdx.y + blockIdx.y * blockDim.y + dev_grid.guard[1] - shift;
  int k =
      threadIdx.z + blockIdx.z * blockDim.z + dev_grid.guard[2] - shift;
  if (i < dev_grid.dims[0] - dev_grid.guard[0] + shift &&
      j < dev_grid.dims[1] - dev_grid.guard[1] + shift &&
      k < dev_grid.dims[2] - dev_grid.guard[2] + shift) {
    ijk = i + j * dev_grid.dims[0] +
          k * dev_grid.dims[0] * dev_grid.dims[1];
    Scalar x0 = dev_grid.pos(0, i, 1);
    Scalar y0 = dev_grid.pos(1, j, 1);
    Scalar z0 = dev_grid.pos(2, k, 1);
    Scalar rl = 2.0 * dev_params.radius;
    // Smoothing scale
    Scalar scaleEpar = 1.0 * dev_grid.delta[0];
    Scalar scaleEperp = 0.5 * dev_grid.delta[0];
    Scalar scaleBperp = scaleEpar;
    Scalar scaleBpar = scaleBperp;
    Scalar d1 = 4 * dev_grid.delta[0];
    Scalar d0 = 0;
    Scalar phase = dev_params.omega * t;

    if (x0 * x0 + y0 * y0 + z0 * z0 < rl * rl) {
      Scalar w = dev_params.omega;

      // set Ex
      x = dev_grid.pos(0, i, 0);
      y = dev_grid.pos(1, j, 1);
      z = dev_grid.pos(2, k, 1);
      r2 = x * x + y * y + z * z;
      r = std::sqrt(r2);
      vx = -w * y;
      vy = w * x;
      intex = interpolate(Ex, ijk, Stagger(0b110), Stagger(0b110),
                          dev_grid.dims[0], dev_grid.dims[1]);
      intey = interpolate(Ey, ijk, Stagger(0b101), Stagger(0b110),
                          dev_grid.dims[0], dev_grid.dims[1]);
      intez = interpolate(Ez, ijk, Stagger(0b011), Stagger(0b110),
                          dev_grid.dims[0], dev_grid.dims[1]);
      intbx = interpolate(Bx, ijk, Stagger(0b001), Stagger(0b110),
                          dev_grid.dims[0], dev_grid.dims[1]);
      intby = interpolate(By, ijk, Stagger(0b010), Stagger(0b110),
                          dev_grid.dims[0], dev_grid.dims[1]);
      intbz = interpolate(Bz, ijk, Stagger(0b100), Stagger(0b110),
                          dev_grid.dims[0], dev_grid.dims[1]);
      exn = -vy * intbz;
      eyn = vx * intbz;
      ezn = -vx * intby + vy * intbx;
      s = shape(r, dev_params.radius - d0, scaleEperp);
      Exnew[ijk] =
          (exn * x + eyn * y + ezn * z) * x / r2 * s +
          (intex * x + intey * y + intez * z) * x / r2 * (1 - s);
      s = shape(r, dev_params.radius - d0, scaleEpar);
      Exnew[ijk] +=
          (exn - (exn * x + eyn * y + ezn * z) * x / r2) * s +
          (intex - (intex * x + intey * y + intez * z) * x / r2) *
              (1 - s);

      // set Ey
      x = dev_grid.pos(0, i, 1);
      y = dev_grid.pos(1, j, 0);
      z = dev_grid.pos(2, k, 1);
      r2 = x * x + y * y + z * z;
      r = std::sqrt(r2);
      vx = -w * y;
      vy = w * x;
      intex = interpolate(Ex, ijk, Stagger(0b110), Stagger(0b101),
                          dev_grid.dims[0], dev_grid.dims[1]);
      intey = interpolate(Ey, ijk, Stagger(0b101), Stagger(0b101),
                          dev_grid.dims[0], dev_grid.dims[1]);
      intez = interpolate(Ez, ijk, Stagger(0b011), Stagger(0b101),
                          dev_grid.dims[0], dev_grid.dims[1]);
      intbx = interpolate(Bx, ijk, Stagger(0b001), Stagger(0b101),
                          dev_grid.dims[0], dev_grid.dims[1]);
      intby = interpolate(By, ijk, Stagger(0b010), Stagger(0b101),
                          dev_grid.dims[0], dev_grid.dims[1]);
      intbz = interpolate(Bz, ijk, Stagger(0b100), Stagger(0b101),
                          dev_grid.dims[0], dev_grid.dims[1]);
      exn = -vy * intbz;
      eyn = vx * intbz;
      ezn = -vx * intby + vy * intbx;
      s = shape(r, dev_params.radius - d0, scaleEperp);
      Eynew[ijk] =
          (exn * x + eyn * y + ezn * z) * y / r2 * s +
          (intex * x + intey * y + intez * z) * y / r2 * (1 - s);
      s = shape(r, dev_params.radius - d0, scaleEpar);
      Eynew[ijk] +=
          (eyn - (exn * x + eyn * y + ezn * z) * y / r2) * s +
          (intey - (intex * x + intey * y + intez * z) * y / r2) *
              (1 - s);

      // set Ez
      x = dev_grid.pos(0, i, 1);
      y = dev_grid.pos(1, j, 1);
      z = dev_grid.pos(2, k, 0);
      r2 = x * x + y * y + z * z;
      r = std::sqrt(r2);
      vx = -w * y;
      vy = w * x;
      intex = interpolate(Ex, ijk, Stagger(0b110), Stagger(0b011),
                          dev_grid.dims[0], dev_grid.dims[1]);
      intey = interpolate(Ey, ijk, Stagger(0b101), Stagger(0b011),
                          dev_grid.dims[0], dev_grid.dims[1]);
      intez = interpolate(Ez, ijk, Stagger(0b011), Stagger(0b011),
                          dev_grid.dims[0], dev_grid.dims[1]);
      intbx = interpolate(Bx, ijk, Stagger(0b001), Stagger(0b011),
                          dev_grid.dims[0], dev_grid.dims[1]);
      intby = interpolate(By, ijk, Stagger(0b010), Stagger(0b011),
                          dev_grid.dims[0], dev_grid.dims[1]);
      intbz = interpolate(Bz, ijk, Stagger(0b100), Stagger(0b011),
                          dev_grid.dims[0], dev_grid.dims[1]);
      exn = -vy * intbz;
      eyn = vx * intbz;
      ezn = -vx * intby + vy * intbx;
      s = shape(r, dev_params.radius - d0, scaleEperp);
      Eznew[ijk] =
          (exn * x + eyn * y + ezn * z) * z / r2 * s +
          (intex * x + intey * y + intez * z) * z / r2 * (1 - s);
      s = shape(r, dev_params.radius - d0, scaleEpar);
      Eznew[ijk] +=
          (ezn - (exn * x + eyn * y + ezn * z) * z / r2) * s +
          (intez - (intex * x + intey * y + intez * z) * z / r2) *
              (1 - s);
    } else {
      Exnew[ijk] = Ex[ijk];
      Eynew[ijk] = Ey[ijk];
      Eznew[ijk] = Ez[ijk];
    }
  }
}



field_solver::field_solver(sim_data &mydata, sim_environment &env)
    : m_data(mydata), m_env(env) {
  En = vector_field<Scalar>(m_data.env.grid());
  dE = vector_field<Scalar>(m_data.env.grid());
  En.copy_stagger(m_data.E);
  dE.copy_stagger(m_data.E);
  En.initialize();
  dE.initialize();

  Bn = vector_field<Scalar>(m_data.env.grid());
  dB = vector_field<Scalar>(m_data.env.grid());
  Bn.copy_stagger(m_data.B);
  dB.copy_stagger(m_data.B);
  Bn.initialize();
  dB.initialize();

  rho = multi_array<Scalar>(m_data.env.grid().extent());
  rho.assign_dev(0.0);

  blockGroupSize =
      dim3((m_data.env.grid().reduced_dim(0) +
            m_env.params().shift_ghost * 2 + blockSize.x - 1) /
               blockSize.x,
           (m_data.env.grid().reduced_dim(1) +
            m_env.params().shift_ghost * 2 + blockSize.y - 1) /
               blockSize.y,
           (m_data.env.grid().reduced_dim(2) +
            m_env.params().shift_ghost * 2 + blockSize.z - 1) /
               blockSize.z);
  std::cout << blockSize.x << ", " << blockSize.y << ", " << blockSize.z
            << std::endl;
  std::cout << blockGroupSize.x << ", " << blockGroupSize.y << ", "
            << blockGroupSize.z << std::endl;
}

field_solver::~field_solver() {}

void
field_solver::evolve_fields(Scalar t) {
  // RANGE_PUSH("Compute", CLR_GREEN);
  copy_fields();

  // substep #1:
  rk_push();
  rk_update(1.0, 0.0, 1.0);
  clean_epar();
  check_eGTb();
  if (m_env.params().pulsar) boundary_pulsar(t + m_env.params().dt);
  CudaSafeCall(hipDeviceSynchronize());
  // RANGE_POP;
  m_env.send_guard_cells(m_data);

  // substep #2:
  // RANGE_PUSH("Compute", CLR_GREEN);
  rk_push();
  rk_update(0.75, 0.25, 0.25);
  clean_epar();
  check_eGTb();
  if (m_env.params().pulsar) boundary_pulsar(t + 0.5 * m_env.params().dt);
  CudaSafeCall(hipDeviceSynchronize());
  // RANGE_POP;
  m_env.send_guard_cells(m_data);

  // substep #3:
  // RANGE_PUSH("Compute", CLR_GREEN);
  rk_push();
  rk_update(1.0 / 3.0, 2.0 / 3.0, 2.0 / 3.0);
  clean_epar();
  check_eGTb();
  if (m_env.params().pulsar) boundary_pulsar(t + m_env.params().dt);
  boundary_absorbing();
  CudaSafeCall(hipDeviceSynchronize());
  // RANGE_POP;

  m_env.send_guard_cells(m_data);
}

void
field_solver::copy_fields() {
  // `En = E, Bn = B`:
  En.copy_from(m_data.E);
  Bn.copy_from(m_data.B);
  dE.initialize();
  dB.initialize();
}

void
field_solver::rk_push() {
  // `rho = div E`
  // kernel_compute_rho<<<gridSize, blockSize>>>(
  kernel_compute_rho_thread<<<blockGroupSize, blockSize>>>(
      m_data.E.dev_ptr(0), m_data.E.dev_ptr(1), m_data.E.dev_ptr(2),
      rho.dev_ptr(), m_env.params().shift_ghost);
  CudaCheckError();
  // `dE = curl B - curl B0 - j, dB = -curl E`
  // kernel_rk_push<<<g, blockSize>>>(
  kernel_rk_push_thread<<<blockGroupSize, blockSize>>>(
      m_data.E.dev_ptr(0), m_data.E.dev_ptr(1), m_data.E.dev_ptr(2),
      m_data.B.dev_ptr(0), m_data.B.dev_ptr(1), m_data.B.dev_ptr(2),
      m_data.B0.dev_ptr(0), m_data.B0.dev_ptr(1), m_data.B0.dev_ptr(2),
      dE.dev_ptr(0), dE.dev_ptr(1), dE.dev_ptr(2), dB.dev_ptr(0),
      dB.dev_ptr(1), dB.dev_ptr(2), rho.dev_ptr(),
      m_env.params().shift_ghost);
  CudaCheckError();
}

void
field_solver::rk_update(Scalar rk_c1, Scalar rk_c2, Scalar rk_c3) {
  // `E = c1 En + c2 E + c3 dE, B = c1 Bn + c2 B + c3 dB`
  // kernel_rk_update<<<dim3(8, 16, 16), dim3(64, 4, 4)>>>(
  kernel_rk_update_thread<<<blockGroupSize, blockSize>>>(
      m_data.E.dev_ptr(0), m_data.E.dev_ptr(1), m_data.E.dev_ptr(2),
      m_data.B.dev_ptr(0), m_data.B.dev_ptr(1), m_data.B.dev_ptr(2),
      En.dev_ptr(0), En.dev_ptr(1), En.dev_ptr(2), Bn.dev_ptr(0),
      Bn.dev_ptr(1), Bn.dev_ptr(2), dE.dev_ptr(0), dE.dev_ptr(1),
      dE.dev_ptr(2), dB.dev_ptr(0), dB.dev_ptr(1), dB.dev_ptr(2), rk_c1,
      rk_c2, rk_c3, m_env.params().shift_ghost);
  CudaCheckError();
}

void
field_solver::clean_epar() {
  // clean `E || B`
  // kernel_clean_epar<<<gridSize, blockSize>>>(
  kernel_clean_epar_thread<<<blockGroupSize, blockSize>>>(
      m_data.E.dev_ptr(0), m_data.E.dev_ptr(1), m_data.E.dev_ptr(2),
      m_data.B.dev_ptr(0), m_data.B.dev_ptr(1), m_data.B.dev_ptr(2),
      dE.dev_ptr(0), dE.dev_ptr(1), dE.dev_ptr(2),
      m_env.params().shift_ghost);
  CudaCheckError();
}

void
field_solver::check_eGTb() {
  // renormalizing `E > B`
  // kernel_check_eGTb<<<dim3(8, 16, 16), dim3(32, 4, 4)>>>(
  kernel_check_eGTb_thread<<<blockGroupSize, blockSize>>>(
      dE.dev_ptr(0), dE.dev_ptr(1), dE.dev_ptr(2), m_data.E.dev_ptr(0),
      m_data.E.dev_ptr(1), m_data.E.dev_ptr(2), m_data.B.dev_ptr(0),
      m_data.B.dev_ptr(1), m_data.B.dev_ptr(2),
      m_env.params().shift_ghost);
  CudaCheckError();
}

void
field_solver::boundary_pulsar(Scalar t) {
  kernel_boundary_pulsar_B_thread<<<blockGroupSize, blockSize>>>(
      m_data.B.dev_ptr(0), m_data.B.dev_ptr(1), m_data.B.dev_ptr(2),
      dB.dev_ptr(0), dB.dev_ptr(1), dB.dev_ptr(2), t,
      m_env.params().shift_ghost);
  CudaCheckError();
  m_data.B.copy_from(dB);
  kernel_boundary_pulsar_E_thread<<<blockGroupSize, blockSize>>>(
      m_data.E.dev_ptr(0), m_data.E.dev_ptr(1), m_data.E.dev_ptr(2),
      m_data.B.dev_ptr(0), m_data.B.dev_ptr(1), m_data.B.dev_ptr(2),
      dE.dev_ptr(0), dE.dev_ptr(1), dE.dev_ptr(2), t,
      m_env.params().shift_ghost);
  CudaCheckError();
  m_data.E.copy_from(dE);
}

void
field_solver::boundary_absorbing() {
  kernel_boundary_absorbing_thread<<<blockGroupSize, blockSize>>>(
      En.dev_ptr(0), En.dev_ptr(1), En.dev_ptr(2), Bn.dev_ptr(0),
      Bn.dev_ptr(1), Bn.dev_ptr(2), m_data.E.dev_ptr(0),
      m_data.E.dev_ptr(1), m_data.E.dev_ptr(2), m_data.B.dev_ptr(0),
      m_data.B.dev_ptr(1), m_data.B.dev_ptr(2),
      m_env.params().shift_ghost);
  CudaCheckError();
}

Scalar
field_solver::total_energy(vector_field<Scalar> &f) {
  f.sync_to_host();
  Scalar Wtmp = 0.0, W = 0.0;
  Scalar xh = m_env.params().lower[0] + m_env.params().size[0] -
              m_env.params().pml[0] * m_env.grid().delta[0];
  Scalar xl = m_env.params().lower[0] +
              m_env.params().pml[0] * m_env.grid().delta[0];
  Scalar yh = m_env.params().lower[1] + m_env.params().size[1] -
              m_env.params().pml[1] * m_env.grid().delta[1];
  Scalar yl = m_env.params().lower[1] +
              m_env.params().pml[1] * m_env.grid().delta[1];
  Scalar zh = m_env.params().lower[2] + m_env.params().size[2] -
              m_env.params().pml[2] * m_env.grid().delta[2];
  Scalar zl = m_env.params().lower[2] +
              m_env.params().pml[2] * m_env.grid().delta[2];
  for (int k = m_env.grid().guard[2];
       k < m_env.grid().dims[2] - m_env.grid().guard[2]; ++k) {
    for (int j = m_env.grid().guard[1];
         j < m_env.grid().dims[1] - m_env.grid().guard[1]; ++j) {
      for (int i = m_env.grid().guard[0];
           i < m_env.grid().dims[0] - m_env.grid().guard[0]; ++i) {
        int ijk = i + j * m_env.grid().dims[0] +
                  k * m_env.grid().dims[0] * m_env.grid().dims[1];
        Scalar x = m_env.grid().pos(0, i, 1);
        Scalar y = m_env.grid().pos(1, j, 1);
        Scalar z = m_env.grid().pos(2, k, 1);
        Scalar r = std::sqrt(x * x + y * y + z * z);
        Scalar fx = interpolate(f.host_ptr(0), ijk, f.stagger(0), Stagger(0b111),
                          m_env.grid().dims[0], m_env.grid().dims[1]);
        Scalar fy = interpolate(f.host_ptr(1), ijk, f.stagger(1), Stagger(0b111),
                          m_env.grid().dims[0], m_env.grid().dims[1]);
        Scalar fz = interpolate(f.host_ptr(2), ijk, f.stagger(2), Stagger(0b111),
                          m_env.grid().dims[0], m_env.grid().dims[1]);
        if ((!(m_env.params().pulsar && r < m_env.params().radius)) && x < xh &&
            x > xl && y < yh && y > yl && z < zh && z > zl)  {
          Wtmp += fx * fx + fy * fy + fz * fz;
        }
      }
    }
  }
  MPI_Reduce(&Wtmp, &W, 1, m_env.scalar_type(), MPI_SUM, 0,
             m_env.world());
  return W;
}

}  // namespace Coffee
